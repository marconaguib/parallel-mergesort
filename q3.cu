#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BILLION  1000000000.0F

// Pour pouvoir experimenter les performances avec les différents types
// FMT  Permet d'avoir un % adapté pour le printf et donc de pas avoir de warning
#define TYPE int
#define FMT  "d"

typedef struct
{
   int x ;
   int y ;
}  Point ;

void MergeSimpleHOST(TYPE *A, TYPE *B, TYPE *M, int cardA, int cardB)
{
    int j = 0;
    int i = 0;

    // On utilise comme pointeurs les arguments de la fonctions
    while (i + j < cardA + cardB )
    {
        if (i >= cardA ) // On a épuisé tout les elts de A, donc on complete avec B
        {  *M = *B; // on utilise les pointeurs pour éviter de faire l opération i+j et se déplacer  = gain de performance
            M = M + 1 ; // Je déplace les pointeurs
            B = B + 1 ;
            j = j + 1 ;
        }
        else if ((j >= cardB) || (*A < *B))
        {   *M = *A ; M = M + 1 ; A = A + 1 ; i = i + 1 ; }
        else
        {   *M = *B ; M = M + 1 ; B = B + 1 ; j = j + 1 ; }
    }
} // End of MergeSimpleHOST

__global__ void MergeSmall_k(TYPE *A, TYPE *B, TYPE *M, int sizeA, int sizeB)
{
    int i = threadIdx.x ;
    Point K, P, Q;
    int offset ;

    if (i >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements
    if ((sizeA == 0) || (sizeB == 0)) { return ; } // Un vecteur "NULL", donc l'autre est trie par hypothese

    if (i > sizeA)
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ;
         Q.x = K.x + offset ; Q.y = K.y - offset ;

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ;
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

}

__global__ void PathBig(TYPE * CudaVecteurA, TYPE * CudaVecteurB, int sizeA , int sizeB, int * CudaDiagBx, int * CudaDiagAy, int nbthread,int NbWindows)
{
    // Initiaise les diagonales
    if(threadIdx.x == 0)
      {
      CudaDiagBx[0] = CudaDiagAy[0] = 0 ;
      CudaDiagBx[NbWindows] = sizeB ;
      CudaDiagAy[NbWindows] = sizeA ;
      }
    int nth = threadIdx.x; // On explore le nth diagonale
    Point K, P, Q ;
    int   px , py ;
    TYPE * A = CudaVecteurA ;
    TYPE * B = CudaVecteurB ;
    int offset ;
    int numDiag  = (nth+1) * nbthread -1 ; // Les tableaux vont de 0 à N-1
	  if (numDiag > sizeA)
        {
    		K.x = numDiag - sizeA ; K.y = sizeA ;
    		P.x = sizeA ; P.y = numDiag - sizeA ;
    	}
    	else // x ~ horizontal
        {
    		K.x = 0 ; K.y = numDiag ;
    		P.x = numDiag ; P.y = 0 ;
    	}
    	while (1)
        {
             offset = abs(K.y - P.y) / 2 ;
             Q.x = K.x + offset ; Q.y = K.y - offset ;

             if ( (Q.y >= 0) && (Q.x <= sizeB) &&
                  ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
             {
                  if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
                  {
                       px = Q.x ; py = Q.y ;
                       if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                       {  // v = A[Q.y] ;
                          py ++ ;
                       }
                       else
                       {  // v = B[Q.x] ;
                          px ++ ;
                       }
                       // printf("Analyse Diagonale Point de Sortie ref %d - M %" FMT " Q (A Q.y %d) (B Q.x %d) rv.x %d rv.y %d\n",i,v,Q.y,Q.x,rv->x,rv->y) ;
                       CudaDiagBx[nth+1] = px ; CudaDiagAy[nth+1] = py ;
                       break ; // Pour simuler passage au thread suivant
                  }
                  else
                  {  K.x = Q.x + 1 ; K.y = Q.y - 1 ;  }
             }
             else
             { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    	}
} // End of PathBig

__global__ void MergeBig_k(TYPE * CudaVecteurA, TYPE * CudaVecteurB, TYPE * CudaVecteurC, int * CudaDiagAy, int * CudaDiagBx , int nbthread)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // On range le ieme elet
    int diag = (i / nbthread)  ;   // Dans quel fenêtre est-il ?
    int indC = nbthread * diag ;

    TYPE *A = CudaVecteurA+CudaDiagAy[diag] ;
    TYPE *B = CudaVecteurB+CudaDiagBx[diag] ;
    TYPE *M = CudaVecteurC + indC  ;
    int sizeA = CudaDiagAy[diag+1]-CudaDiagAy[diag] ;
    int sizeB = CudaDiagBx[diag+1]-CudaDiagBx[diag] ;

    Point K, P, Q;
    int offset ;

    i = i % nbthread ; // On recadre i dans le nouvel espace
    if (i >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements
    if (i > sizeA)
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ;
         Q.x = K.x + offset ; Q.y = K.y - offset ;

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ;
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

}

void Affiche (char * tabMsg, TYPE * ptBuffer, int NB)
{
   TYPE * pt = ptBuffer ;
   for ( int k = 0 ; k < NB  ; k++ , pt ++)
   {   printf(" - %s[%03d] = %6" FMT, tabMsg, k , *pt) ;
       if ((k % 5) == (4))
       {  printf("\n") ; fflush(stdout);  }

   }
   printf("\n") ;
  fflush(stdout);
}

int check(char * msg, int Nb, TYPE * pto)
{
    TYPE * pt1 = pto ;
    TYPE * pt2 = pto + 1 ;
    int i ;

    for (i = 0 ; i < Nb-1 ; i ++)
    {
        if (*pt1 > *pt2)
        { printf("Check %s pour %d - Erreur en position %d %" FMT " > %" FMT " \n", msg, Nb, i, *pt1, *pt2) ;
    //	return i ;
	exit(25) ;
        }
        pt1 ++ ; pt2 ++ ;
    }

    printf("Check %s pour %d est OK \n", msg, Nb) ;
    return 0 ;
}

void MergeSort(TYPE * M, int sizeM)
{

    //Declarations
    hipError_t errCuda;
    TYPE * ptori  = NULL ; // pointeur origine
    TYPE * ptdest = NULL ; // pointeur destination
    TYPE * pttmp ;

    TYPE * cudaOri  = NULL ; // pointeur orgine dans CUDA
    TYPE * cudaDest = NULL ; // pointeur dest dans CUDA

    int t ;
    int  * CudaDiagBx   = NULL ;
    int  * CudaDiagAy   = NULL ;

    //Allocation
    if ((ptdest = (TYPE *) malloc(sizeM * sizeof(TYPE))) == NULL)
        { printf("PB allocation VecteurM2n") ; exit (1) ; }

    if (hipSuccess != (errCuda = hipMalloc((void**)&cudaOri, sizeM * sizeof(TYPE))))
        { printf("PB allocation CudaOri - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    if (hipSuccess != (errCuda = hipMalloc((void**)&cudaDest, sizeM * sizeof(TYPE))))
        { printf("PB allocation CudaDest - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    if (hipSuccess != (errCuda = hipMalloc((void**)&CudaDiagBx, (1025 + 1) * sizeof(int))))
        { printf("PB allocation CudaDiagBx pour  - %d - %s \n", errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

     if (hipSuccess != (errCuda = hipMalloc((void**)&CudaDiagAy, (1025 + 1)* sizeof(int))))
         { printf("PB allocation CudaDiagAy - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout);  exit (1) ; }

    ptori  = M ;

    // Première itération on le trie à la main pour gagner du temps
    for (int i = 0 ; i < sizeM ; i += 2 )
    {
        if (ptori[i] > ptori[i+1])
        {
            ptdest[i+1] = ptori[i];
            ptdest[i]   = ptori[i+1] ;
        }
	      else
	      {
            ptdest[i]   = ptori[i];
            ptdest[i+1] = ptori[i+1];
      	}
    }

    // Flip Flop entre ptori et ptdest
    pttmp = ptdest ;
    ptdest= ptori ;
    ptori = pttmp ;

    t=2;
    // Seconde itération on le fait en séquentiel avec l'algo A du sujet pour gagner du temps
    for (int i = 0 ; i < sizeM ; i = i+(2*t))
    {
        int sizeA = min(t,sizeM-i);
        int sizeB = min(t,max(sizeM-(i+t),0));
        TYPE * ptA = ptori + i;
        TYPE * ptB = ptori + i + sizeA ;
        TYPE * ptM = ptdest + i ;
        MergeSimpleHOST(ptA, ptB, ptM, sizeA , sizeB) ;
      }

    if (hipSuccess != (errCuda = hipMemcpy(cudaOri, ptdest, sizeM * sizeof(TYPE), hipMemcpyHostToDevice)))
    { printf("PB Copie Host ptDest -> cudaOri - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    //Le reste des itérations on utilise mergesmall et mergebig
    for ( t = 4 ; t < sizeM ; t= t*2)
    {
        for ( int i = 0 ; i < sizeM ; i = i + (2*t))
        {
            int sizeA = min(t,sizeM-i);
            int sizeB = min(t,max(sizeM-(i+t),0));

	          TYPE * CudaVecteurA = cudaOri + i ;
	          TYPE * CudaVecteurB = cudaOri + i + sizeA ;

	          if ((sizeA == 0) || (sizeB == 0))
	          {
                if (sizeA != 0)
                {
                   if (hipSuccess != (errCuda = hipMemcpy(cudaDest + i, CudaVecteurA , sizeA * sizeof(TYPE), hipMemcpyDeviceToDevice)))
                   { printf("PB Copie Cuda A -> ptDes rab %d - %d - %s \n",sizeA, errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }
		            }
                continue ;
	          }

            //Merge
            if (sizeA+sizeB <= 1024)
            {
                MergeSmall_k<<<1,sizeA+sizeB>>> (CudaVecteurA,CudaVecteurB,cudaDest+i,sizeA,sizeB);
            }
            else
            {
	        	    int nbthread = 1024;
                int NbDiagonale  = (sizeA + sizeB) / nbthread ;
		            if (NbDiagonale > 1024)
		            { printf("Oups, on n'a pas fait le code pour nbDiag %d > 1024\n",NbDiagonale) ;
		              return ;
		            }
                int NbWindows    = NbDiagonale ;
                NbWindows   += (((sizeA + sizeB) % nbthread) == 0)?0:1 ;  // si (SizeA + SizeB) % nbthread == 0 alors nbWindows = 0  sinon = 1
                PathBig<<<1,NbDiagonale>>>(CudaVecteurA, CudaVecteurB, sizeA , sizeB, CudaDiagBx, CudaDiagAy, nbthread,NbWindows) ;
                int nbBlock  = (sizeA+sizeB) / 1024 ;
       	        nbBlock += ((sizeA+sizeB) % 1024)?1:0 ;
       	        MergeBig_k<<<nbBlock,1024>>> (CudaVecteurA, CudaVecteurB, cudaDest+i, CudaDiagAy, CudaDiagBx, nbthread) ;
            }
          }// End for i
	    // Flip Flop entre les bancs cudaOri et cudaDest
	    TYPE * cudaTmp = cudaDest ;
	    cudaDest = cudaOri ;
	    cudaOri  = cudaTmp ;
    } // End of loop t

    if (hipSuccess != hipMemcpy(M, cudaOri, sizeM * sizeof(TYPE), hipMemcpyDeviceToHost))
    { printf("PB copie cuda M -> host M \n") ; fflush(stdout);  exit(2) ; }

    // Free
    if (cudaOri != NULL) { hipFree(cudaOri) ; cudaOri = NULL ; }
    if (cudaDest != NULL) { hipFree(cudaDest) ; cudaDest = NULL ; }
    if (CudaDiagAy != NULL) { hipFree(CudaDiagAy) ; CudaDiagAy = NULL ; }
    if (CudaDiagBx != NULL) { hipFree(CudaDiagBx) ; CudaDiagBx = NULL ; }
}

int main(int argc, char ** argv)
{
    //déclaration
    int sizeM;
    TYPE* M;
    hipEvent_t Start; hipEvent_t Stop; hipEventCreate(&Start) ; hipEventCreate(&Stop) ;
    float temps_iter;
    srand(1998);
    for (int i=0; i<11; i++)
    {
        temps_iter=0;
        sizeM = (1024*pow(2,i));
        M = (TYPE *) malloc(sizeM * sizeof(TYPE));
        for (int i =0; i<sizeM; i++) M[i]=rand();

        hipEventRecord(Start);
        MergeSort(M,sizeM);
        hipEventRecord(Stop) ;

        check("tableau M après", sizeM, M);
        hipEventElapsedTime(&temps_iter, Start, Stop) ;

        free(M);
        printf("Duree pour %d : %f s\n",sizeM,temps_iter/1000) ;
    }

    return 0 ;
}
