#include "hip/hip_runtime.h"
// **********************************************************************************************************************************************
// KERNELS PRINCIPAUX POUR PROJET HPC
// ASTRID LEGAY ET MARCO NAGUIB - MAIN 5
// 15 DECEMBRE 2020
// **********************************************************************************************************************************************

// STRUCTURE UTILE
// Pour utiliser la notion de point
typedef struct
{
   int x ;
   int y ;
}  Point ;

// **********************************************************************************************************************************************
//  QUESTIONS 1
// **********************************************************************************************************************************************

// MergeSmall_k permet de merger le tableau A et B (déjà triés) dans M
// On prend en entrée le tableau A trié, le tableau B trié, le tableau M pour mettre le résultat, la taille de A et la taille de B
// Parallélisation de l'algorithme B
__global__ void MergeSmall_k(TYPE *A, TYPE *B, TYPE *M, int cardA, int cardB)
{
    Point K;
    Point P;
    Point Q;
    int offset ;
    int i = threadIdx.x ; // Id du thread, permet de savoir quelle valeur va être rangé à sa place définitive.
    {
    	if (i > cardA)
      {
    		K.x = i - cardA ; K.y = cardA ;
    		P.x = cardA ; P.y = i - cardA ;
    	}
    	else
      {
    		K.x = 0 ; K.y = i ;
    		P.x = i ; P.y = 0 ;
    	}
    	while (1)
      {
        offset = abs (K.y - P.y) / 2 ;
        Q.x = K.x + offset ; Q.y = K.y - offset ; // Q est bien sur une diagonale à 45°

    		if (((Q.y >= 0 ) && (Q.x <= cardB)) &&
    			((Q.y == cardA) || (Q.x == 0) || (A[Q.y]>B[Q.x -1]))){

    			if ((Q.x == cardB) || (Q.y == 0) || (A[Q.y-1]<=B[Q.x]))
          {
    				if((Q.y < cardA) && ((Q.x == cardB) || (A[Q.y] <= B[Q.x])))
            {  M[i]= A[Q.y] ; }
    				else
            {	 M[i] = B[Q.x] ; }
    				break;  // Pour simuler passage au thread suivant
    			}
    			else
          {  K.x = Q.x +1 ; K.y = Q.y - 1 ; }
    		}
    		else
        {	P.x = Q.x -1 ; P.y = Q.y +1 ; }
    	}
    }
}

// Ajout des lignes 68 à 82 pour travailler sur la mémoire shared
__global__ void MergeSmallShared_k(TYPE *GlobalCudaA, TYPE *GlobalCudaB, TYPE *M, int sizeA, int sizeB)
{
    extern __shared__ TYPE dataAB[] ; // j utilise la mémoire partagée entre les threads
    unsigned int tid = threadIdx.x; // numéro du thread dans le block courant
    unsigned int i   = blockIdx.x*blockDim.x + threadIdx.x; // numéro du thread  sur l ensemble des blocks

    if (tid >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements

 // Chargement des données dans la mémoire partagée par le thread ;
     dataAB[tid] = (i < sizeA)?GlobalCudaA[i]:GlobalCudaB[i-sizeA] ;  // voir si on travaille sur le vecteur A ou B

 // On attend que tous les threads aient faits le travail ( chargé la mémoire)
    __syncthreads();  //

 // On recadre nos pointeurs pourqu'ils pointent vers la mémoire partagée et non la globale
    TYPE * A = dataAB ;
    TYPE * B = dataAB + sizeA ;

    Point K, P, Q;
    int offset ;

    if (i > sizeA)
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ;
         Q.x = K.x + offset ; Q.y = K.y - offset ;

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ;
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

}

// **********************************************************************************************************************************************
// QUESTION 2
// **********************************************************************************************************************************************


__global__ void PathBig(TYPE * CudaVecteurA, TYPE * CudaVecteurB, int sizeA , int sizeB, int * CudaDiagBx, int * CudaDiagAy, int nbthread, int NbWindows)
{
    // A : an array of size sizeA
    // B : an array of size sizeB
    // (CudaDiagBx,CudaDiagAy) recieve the respective coordinates of the "red points"
    // nbthread : Number of threads, preferably 1024
    // NbWindows : Number of windows

    //Initialisation diagolane
    CudaDiagBx[0] = CudaDiagAy[0] = 0 ; //(0,0)
    CudaDiagBx[NbWindows] = sizeB ;
    CudaDiagAy[NbWindows] = sizeA ; //(sizeA,sizeB)

    int nth = threadIdx.x; // On explore le nth diagonale
    Point K, P, Q ;
    int   px , py ;
    TYPE * A = CudaVecteurA ;
    TYPE * B = CudaVecteurB ;
    int offset ;
    int numDiag  = (nth+1) * nbthread -1 ; // Les tableaux vont de 0 à N-1
	  if (numDiag > sizeA)
        {
    		K.x = numDiag - sizeA ; K.y = sizeA ;
    		P.x = sizeA ; P.y = numDiag - sizeA ;
    	}
    	else // x ~ horizontal
        {
    		K.x = 0 ; K.y = numDiag ;
    		P.x = numDiag ; P.y = 0 ;
    	}
    	while (1)
        {
             offset = abs(K.y - P.y) / 2 ;
             Q.x = K.x + offset ; Q.y = K.y - offset ;

             if ( (Q.y >= 0) && (Q.x <= sizeB) &&
                  ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
             {
                  if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
                  {
                       px = Q.x ; py = Q.y ;
                       if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                       {  // v = A[Q.y] ;
                          py ++ ;
                       }
                       else
                       {  // v = B[Q.x] ;
                          px ++ ;
                       }
                       // printf("Analyse Diagonale Point de Sortie ref %d - M %" FMT " Q (A Q.y %d) (B Q.x %d) rv.x %d rv.y %d\n",i,v,Q.y,Q.x,rv->x,rv->y) ;
                       CudaDiagBx[nth+1] = px ; CudaDiagAy[nth+1] = py ;
                       break ; // Pour simuler passage au thread suivant
                  }
                  else
                  {  K.x = Q.x + 1 ; K.y = Q.y - 1 ;  }
             }
             else
             { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    	}
}

__global__ void MergeBig_k(TYPE * CudaVecteurA, TYPE * CudaVecteurB, TYPE * CudaVecteurC, int * CudaDiagAy, int * CudaDiagBx , int nbthread)
{

    // int i = threadIdx.x ;     // On renge le Ieme element
    int i = blockIdx.x * blockDim.x + threadIdx.x; // On range le ieme elet
    int diag = (i / nbthread)  ;   // Dans quel fenêtre est-il ?
    int indC = nbthread * diag ;

    TYPE *A = CudaVecteurA+CudaDiagAy[diag] ;
    TYPE *B = CudaVecteurB+CudaDiagBx[diag] ;
    TYPE *M = CudaVecteurC + indC  ;
    int sizeA = CudaDiagAy[diag+1]-CudaDiagAy[diag] ;
    int sizeB = CudaDiagBx[diag+1]-CudaDiagBx[diag] ;

    Point K, P, Q;
    int offset ;

    i = i % nbthread ; // On recadre i dans le nouvel espace
    if (i >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements
    if (i > sizeA)
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ;
         Q.x = K.x + offset ; Q.y = K.y - offset ;

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ;
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

}

// **********************************************************************************************************************************************
// QUESTION  3
// **********************************************************************************************************************************************

// Nous vous avons expliquer l'algortihme de Merge Sort précédement, donc nous travaillons par taille  : t = t*2
// Tout d'abord, il est important de noter qu'on ne travaille en parallèle qu'à partir de la taille 4 pour optimiser le code
// Taille 1 : tri à la "main" sur le HOST
// Taille 2 : tri de l'algorithme A de l'anoncé sur le HOST
// A partir de la taille 4 : mise en place de CUDA sur GPU pour paralléliser : si size A + sizeB <= 1024 : appelle MergeSmall sinon PathBig et MergeBig
// Ensuite nous avons une notion de FLIP/FLOP : mis en place pour éviter de nombreuses copies et ainsi gagner du temps
// Concernant la notion de FLIP/FLOP, je vais l'expliquer avec un schéma avant de monter sur le code

void MergeSort(TYPE * M, int sizeM)
{

    //Declarations
    hipError_t errCuda;
    TYPE * ptori  = NULL ; // pointeur origine
    TYPE * ptdest = NULL ; // pointeur destination
    TYPE * pttmp ;

    TYPE * cudaOri  = NULL ; // pointeur orgine dans CUDA
    TYPE * cudaDest = NULL ; // pointeur dest dans CUDA

    int t ;
    int  * CudaDiagBx   = NULL ;
    int  * CudaDiagAy   = NULL ;

    //Allocation
    if ((ptdest = (TYPE *) malloc(sizeM * sizeof(TYPE))) == NULL)
        { printf("PB allocation VecteurM2n") ; exit (1) ; }

    if (hipSuccess != (errCuda = hipMalloc((void**)&cudaOri, sizeM * sizeof(TYPE))))
        { printf("PB allocation CudaOri - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    if (hipSuccess != (errCuda = hipMalloc((void**)&cudaDest, sizeM * sizeof(TYPE))))
        { printf("PB allocation CudaDest - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    if (hipSuccess != (errCuda = hipMalloc((void**)&CudaDiagBx, (1025 + 1) * sizeof(int))))
        { printf("PB allocation CudaDiagBx pour  - %d - %s \n", errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

     if (hipSuccess != (errCuda = hipMalloc((void**)&CudaDiagAy, (1025 + 1)* sizeof(int))))
         { printf("PB allocation CudaDiagAy - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout);  exit (1) ; }

    ptori  = M ;

    // Première itération on le trie à la main pour gagner du temps
    for (int i = 0 ; i < sizeM ; i += 2 )
    {
        if (ptori[i] > ptori[i+1])
        {
            ptdest[i+1] = ptori[i];
            ptdest[i]   = ptori[i+1] ;
        }
	      else
	      {
            ptdest[i]   = ptori[i];
            ptdest[i+1] = ptori[i+1];
      	}
    }

    // Flip Flop entre ptori et ptdest
    pttmp = ptdest ;
    ptdest= ptori ;
    ptori = pttmp ;

    t=2;
    // Seconde itération on le fait en séquentiel avec l'algo A du sujet pour gagner du temps
    for (int i = 0 ; i < sizeM ; i = i+(2*t))
    {
        int sizeA = min(t,sizeM-i);
        int sizeB = min(t,max(sizeM-(i+t),0));
        TYPE * ptA = ptori + i;
        TYPE * ptB = ptori + i + sizeA ;
        TYPE * ptM = ptdest + i ;
        MergeSimpleHOST(ptA, ptB, ptM, sizeA , sizeB) ;
      }

    if (hipSuccess != (errCuda = hipMemcpy(cudaOri, ptdest, sizeM * sizeof(TYPE), hipMemcpyHostToDevice)))
    { printf("PB Copie Host ptDest -> cudaOri - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    //Le reste des itérations on utilise mergesmall et mergebig
    for ( t = 4 ; t < sizeM ; t= t*2)
    {
        for ( int i = 0 ; i < sizeM ; i = i + (2*t))
        {
            int sizeA = min(t,sizeM-i);
            int sizeB = min(t,max(sizeM-(i+t),0));

	          TYPE * CudaVecteurA = cudaOri + i ;
	          TYPE * CudaVecteurB = cudaOri + i + sizeA ;

	          if ((sizeA == 0) || (sizeB == 0))
	          {
                if (sizeA != 0)
                {
                   if (hipSuccess != (errCuda = hipMemcpy(cudaDest + i, CudaVecteurA , sizeA * sizeof(TYPE), hipMemcpyDeviceToDevice)))
                   { printf("PB Copie Cuda A -> ptDes rab %d - %d - %s \n",sizeA, errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }
		            }
                continue ;
	          }

            //Merge
            if (sizeA+sizeB <= 1024)
            {
                MergeSmall_k<<<1,sizeA+sizeB>>> (CudaVecteurA,CudaVecteurB,cudaDest+i,sizeA,sizeB);
            }
            else
            {
	        	    int nbthread = 1024;
                int NbDiagonale  = (sizeA + sizeB) / nbthread ;
		            if (NbDiagonale > 1024)
		            { printf("Oups, on n'a pas fait le code pour nbDiag %d > 1024\n",NbDiagonale) ;
		              return ;
		            }
                int NbWindows    = NbDiagonale ;
                NbWindows   += (((sizeA + sizeB) % nbthread) == 0)?0:1 ;  // si (SizeA + SizeB) % nbthread == 0 alors nbWindows = 0  sinon = 1
                PathBig<<<1,NbDiagonale>>>(CudaVecteurA, CudaVecteurB, sizeA , sizeB, CudaDiagBx, CudaDiagAy, nbthread,NbWindows) ;
                int nbBlock  = (sizeA+sizeB) / 1024 ;
       	        nbBlock += ((sizeA+sizeB) % 1024)?1:0 ;
       	        MergeBig_k<<<nbBlock,1024>>> (CudaVecteurA, CudaVecteurB, cudaDest+i, CudaDiagAy, CudaDiagBx, nbthread) ;
            }
          }// End for i
	    // Flip Flop entre les bancs cudaOri et cudaDest
	    TYPE * cudaTmp = cudaDest ;
	    cudaDest = cudaOri ;
	    cudaOri  = cudaTmp ;
    } // End of loop t

    if (hipSuccess != hipMemcpy(M, cudaOri, sizeM * sizeof(TYPE), hipMemcpyDeviceToHost))
    { printf("PB copie cuda M -> host M \n") ; fflush(stdout);  exit(2) ; }

    // Free
    if (cudaOri != NULL) { hipFree(cudaOri) ; cudaOri = NULL ; }
    if (cudaDest != NULL) { hipFree(cudaDest) ; cudaDest = NULL ; }
    if (CudaDiagAy != NULL) { hipFree(CudaDiagAy) ; CudaDiagAy = NULL ; }
    if (CudaDiagBx != NULL) { hipFree(CudaDiagBx) ; CudaDiagBx = NULL ; }
}

// **********************************************************************************************************************************************
// QUESTION 5
// **********************************************************************************************************************************************

__global__ void MergeSmallBatch_k(TYPE *ABAB, int sizeM_tot, TYPE* MM, int d)
{
    int i = threadIdx.x%d;
    int Qt = (threadIdx.x-i)/d;
    int gbx = Qt + blockIdx.x*(blockDim.x/d);
    if (threadIdx.x + blockIdx.x*blockDim.x >= sizeM_tot) return;

    int t = d/2;
    int sizeA = t;
    int sizeB = t;

    ABAB=ABAB+gbx*d;
    TYPE* A=ABAB;
    TYPE* B=A+sizeA;
    TYPE* M=MM+gbx*d;


    Point K, P, Q;
    int offset ;

    if (i > sizeA)
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ;
         Q.x = K.x + offset ; Q.y = K.y - offset ;

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ;
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }
}

// **********************************************************************************************************************************************
// PARTIE 3
// **********************************************************************************************************************************************

void MergeSort(TYPE * M, int sizeM)
{
    //Declarations
    hipError_t errCuda;
    TYPE * cudaOri  = NULL ; // pointeur orgine dans CUDA
    TYPE * cudaDest = NULL ; // pointeur dest dans CUDA

    int  * CudaDiagBx   = NULL ;
    int  * CudaDiagAy   = NULL ;

    int t ;

    //Allocation
    if (hipSuccess != (errCuda = hipMalloc((void**)&cudaOri, sizeM * sizeof(TYPE))))
        { printf("PB allocation CudaVecteurM1 - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }  // cleanup a rajouter pour plus propre

    if (hipSuccess != (errCuda = hipMalloc((void**)&cudaDest, sizeM * sizeof(TYPE))))
        { printf("PB allocation CudaVecteurM2 - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    if (hipSuccess != (errCuda = hipMalloc((void**)&CudaDiagBx, 1026 * sizeof(int))))
       { printf("PB allocation CudaDiagBx %d - %d - %s \n", errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    if (hipSuccess != (errCuda = hipMalloc((void**)&CudaDiagAy, 1026 * sizeof(int))))
       { printf("PB allocation CudaDiagAy - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout);  exit (1) ; }

    //Initialiser cudaOri
    if (hipSuccess != (errCuda = hipMemcpy(cudaOri, M, sizeM * sizeof(TYPE), hipMemcpyHostToDevice)))
       { printf("PB Copie Host ptDest -> cudaOri - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    //Trier cudaOri par blocs de 2, puis par blocs de 4, etc jusqua 512
    for ( t = 1 ; t <= 512 and t<sizeM ; t= t*2){
        //partie divisible par d
        int d=t*2;
        int size_AetB = sizeM%d;//taille restante
        MergeSmallBatch_k<<<1024,1024>>>(cudaOri,sizeM-size_AetB,cudaDest,t*2);

        //partie restante
        int sizeA = min(size_AetB,t);
        int sizeB = size_AetB - sizeA;
        TYPE* cudaM = cudaDest+sizeM-size_AetB;
        TYPE* cudaA = cudaOri+sizeM-size_AetB;
        TYPE* cudaB = cudaA+sizeA;
        MergeSmall_k<<<1,sizeA+sizeB>>> (cudaA,cudaB,cudaM,sizeA,sizeB);

        // Flip Flop entre les bancs cudaDest et cudaOri
        TYPE * cudaTmp = cudaDest ;
        cudaDest = cudaOri ;
        cudaOri  = cudaTmp ;
    }

    //t=512 on trie par blocs de taille supérieure à laide de PathBig et MergeBig
    for ( t = t ; t < sizeM ; t= t*2)
    {
        for ( int i = 0 ; i < sizeM ; i = i + (2*t))
        {
            int sizeA = min(t,sizeM-i);
            int sizeB = min(t,max(sizeM-(i+t),0));

            TYPE * CudaVecteurA = cudaOri + i ;
            TYPE * CudaVecteurB = cudaOri + i + sizeA ;

            if ((sizeA == 0) || (sizeB == 0))
            {
                if (sizeA != 0)
                {
                  if (hipSuccess != (errCuda = hipMemcpy(cudaDest + i, CudaVecteurA , sizeA * sizeof(TYPE), hipMemcpyDeviceToDevice)))
                  { printf("PB Copie Cuda A -> ptDes rab %d - %d - %s \n",sizeA, errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }
                }
                continue ;
            }

            //Merge
            if (sizeA+sizeB <= 1024) { printf("Oups, on est censé avoir traité ce cas par le merge batch\n") ; return ;}
            int nbthread = 1024;
            int NbDiagonale  = (sizeA + sizeB) / nbthread ;
            if (NbDiagonale > 1024) { printf("Oups, on n'a pas fait le code pour nbDiag %d > 1024\n",NbDiagonale) ; return ; }
            int NbWindows    =  NbDiagonale ;
            NbWindows   += (((sizeA + sizeB) % nbthread) == 0)?0:1 ;  // si (SizeA + SizeB) % nbthread == 0 alors nbWindows = 0  sinon = 1


            PathBig<<<1,NbDiagonale>>>(CudaVecteurA, CudaVecteurB, sizeA , sizeB, CudaDiagBx, CudaDiagAy, nbthread,NbWindows) ;
            int nbBlock  = (sizeA+sizeB) / 1024 ;
            nbBlock += ((sizeA+sizeB) % 1024)?1:0 ;
            MergeBig_k<<<nbBlock,1024>>> (CudaVecteurA, CudaVecteurB, cudaDest+i, CudaDiagAy, CudaDiagBx, nbthread) ;
        } // End for i

        // Flip Flop entre les bancs cudaDest et cudaOri
        TYPE * cudaTmp = cudaDest ;
        cudaDest = cudaOri ;
        cudaOri  = cudaTmp ;

    }
    //cudaOri est entièrement trié

    //remettre dans M
    if (hipSuccess != hipMemcpy(M, cudaOri, sizeM * sizeof(TYPE), hipMemcpyDeviceToHost))
    { printf("PB copie cuda M -> host M \n") ; fflush(stdout);  exit(2) ; }


    // Free
    if (cudaOri != NULL) { hipFree(cudaOri) ; cudaOri = NULL ; }
    if (cudaDest != NULL) { hipFree(cudaDest) ; cudaDest = NULL ; }
    if (CudaDiagAy != NULL) { hipFree(CudaDiagAy) ; CudaDiagAy = NULL ; }
    if (CudaDiagBx != NULL) { hipFree(CudaDiagBx) ; CudaDiagBx = NULL ; }
}
