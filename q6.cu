#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define BILLION  1000000000.0F

// Pour pouvoir experimenter les performances avec les différents types
// FMT  Permet d'avoir un % adapté pour le printf et donc de pas avoir de warning
#define TYPE int
#define FMT  "d"

typedef struct
{
   int x ;
   int y ;
}  Point ;


__global__ void MergeSmall_k(TYPE *A, TYPE *B, TYPE *M, int sizeA, int sizeB)
{
    int i = threadIdx.x ;
    Point K, P, Q;
    int offset ;

    if (i >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements
    if(sizeA==0) {
        for (int i=0; i<sizeB; i++)
        {
            M[i]=B[i];
        }
        return;
    }
    else if (sizeB==0)
    {
        for (int i=0; i<sizeA; i++)
        {
            M[i]=A[i];
        }
        return;
    }
    if (i > sizeA)
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ;
         Q.x = K.x + offset ; Q.y = K.y - offset ;

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ;
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

}


__global__ void MergeSmallBatch_k(TYPE *M, int sizeM_tot, TYPE* N, int d)
{
    int i = threadIdx.x%d;
    int Qt = (threadIdx.x-i)/d;
    int gbx = Qt + blockIdx.x*(blockDim.x/d);
    if (threadIdx.x + blockIdx.x*blockDim.x >= sizeM_tot) return;

    int t = d/2;
    int sizeA = t;
    int sizeB = t;

    M=M+gbx*d;
    TYPE* A=M;
    TYPE* B=A+sizeA;


    Point K, P, Q;
    int offset ;

    if (i > sizeA)
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ;
         Q.x = K.x + offset ; Q.y = K.y - offset ;

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  N[i+gbx*d] = A[Q.y] ; }
                   else
                   {  N[i+gbx*d] = B[Q.x] ; }
                   break ;
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }
}


__global__ void PathBig(TYPE * CudaVecteurA, TYPE * CudaVecteurB, int sizeA , int sizeB, int * CudaDiagBx, int * CudaDiagAy, int nbthread,int NbWindows)
{
    //init
    if(threadIdx.x==0)
    {
      CudaDiagBx[0] = CudaDiagAy[0] = 0 ;
      CudaDiagBx[NbWindows] = sizeB ;
      CudaDiagAy[NbWindows] = sizeA ;
    }
    int nth = threadIdx.x; // On explore le nth diagonale
    Point K, P, Q ;
    int   px , py ;
    TYPE * A = CudaVecteurA ;
    TYPE * B = CudaVecteurB ;
    int offset ;
    int numDiag  = (nth+1) * nbthread -1 ; // Les tableaux vont de 0 à N-1
	  if (numDiag > sizeA)
        {
    		K.x = numDiag - sizeA ; K.y = sizeA ;
    		P.x = sizeA ; P.y = numDiag - sizeA ;
    	}
    	else // x ~ horizontal
        {
    		K.x = 0 ; K.y = numDiag ;
    		P.x = numDiag ; P.y = 0 ;
    	}
    	while (1)
        {
             offset = abs(K.y - P.y) / 2 ;
             Q.x = K.x + offset ; Q.y = K.y - offset ;

             if ( (Q.y >= 0) && (Q.x <= sizeB) &&
                  ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
             {
                  if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
                  {
                       px = Q.x ; py = Q.y ;
                       if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                       {  // v = A[Q.y] ;
                          py ++ ;
                       }
                       else
                       {  // v = B[Q.x] ;
                          px ++ ;
                       }
                       // printf("Analyse Diagonale Point de Sortie ref %d - M %" FMT " Q (A Q.y %d) (B Q.x %d) rv.x %d rv.y %d\n",i,v,Q.y,Q.x,rv->x,rv->y) ;
                       CudaDiagBx[nth+1] = px ; CudaDiagAy[nth+1] = py ;
                       break ; // Pour simuler passage au thread suivant
                  }
                  else
                  {  K.x = Q.x + 1 ; K.y = Q.y - 1 ;  }
             }
             else
             { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    	}
} // End of PathBig

__global__ void MergeBig_k(TYPE * CudaVecteurA, TYPE * CudaVecteurB, TYPE * CudaVecteurC, int * CudaDiagAy, int * CudaDiagBx , int nbthread)
{
    // int i = threadIdx.x ;     // On renge le Ieme element
    int i = blockIdx.x * blockDim.x + threadIdx.x; // On range le ieme elet
    int diag = (i / nbthread)  ;   // Dans quel fenêtre est-il ?
    int indC = nbthread * diag ;

    TYPE *A = CudaVecteurA+CudaDiagAy[diag] ;
    TYPE *B = CudaVecteurB+CudaDiagBx[diag] ;
    TYPE *M = CudaVecteurC + indC  ;
    int sizeA = CudaDiagAy[diag+1]-CudaDiagAy[diag] ;
    int sizeB = CudaDiagBx[diag+1]-CudaDiagBx[diag] ;

    Point K, P, Q;
    int offset ;

    i = i % nbthread ; // On recadre i dans le nouvel espace
    if (i >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements
    if (i > sizeA)
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ;
         Q.x = K.x + offset ; Q.y = K.y - offset ;

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ;
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

}

void Affiche (char * tabMsg, TYPE * ptBuffer, int NB)
{
   TYPE * pt = ptBuffer ;
   for ( int k = 0 ; k < NB  ; k++ , pt ++)
   {   printf(" - %s[%03d] = %6" FMT, tabMsg, k , *pt) ;
       if ((k % 5) == (4))
       {  printf("\n") ; fflush(stdout);  }

   }
   printf("\n") ;
  fflush(stdout);
}


void AfficheDiag(int * ADiag,int * BDiag, int nb)
{
   for (int i = 0 ; i < nb ; i ++)
   {  printf("Diag[%4d] = (Bx %6d , Ay = %6d)\n",i,BDiag[i], ADiag[i]) ; fflush(stdout);}
    printf("\n") ;
    fflush(stdout);
}

int check(char * msg, int Nb, TYPE * pto)
{
    TYPE * pt1 = pto ;
    TYPE * pt2 = pto + 1 ;
    int i ;

    for (i = 0 ; i < Nb-1 ; i ++)
    {
        if (*pt1 > *pt2)
        {
          printf("Check %s pour %d - Erreur en position %d %" FMT " > %" FMT " \n", msg, Nb, i, *pt1, *pt2) ;
          exit(25) ;
        }
        pt1 ++ ; pt2 ++ ;
    }

    printf("Check %s pour %d est OK \n", msg, Nb) ;
    return 0 ;
}

void MergeSort(TYPE * M, int sizeM)
{
    //Declarations
    hipError_t errCuda;
    TYPE * cudaOri  = NULL ; // pointeur orgine dans CUDA
    TYPE * cudaDest = NULL ; // pointeur dest dans CUDA

    int  * CudaDiagBx   = NULL ;
    int  * CudaDiagAy   = NULL ;

    int t ;

    //Allocation
    if (hipSuccess != (errCuda = hipMalloc((void**)&cudaOri, sizeM * sizeof(TYPE))))
        { printf("PB allocation CudaVecteurM1 - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }  // cleanup a rajouter pour plus propre

    if (hipSuccess != (errCuda = hipMalloc((void**)&cudaDest, sizeM * sizeof(TYPE))))
        { printf("PB allocation CudaVecteurM2 - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    if (hipSuccess != (errCuda = hipMalloc((void**)&CudaDiagBx, 1026 * sizeof(int))))
       { printf("PB allocation CudaDiagBx %d - %d - %s \n", errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    if (hipSuccess != (errCuda = hipMalloc((void**)&CudaDiagAy, 1026 * sizeof(int))))
       { printf("PB allocation CudaDiagAy - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout);  exit (1) ; }

    //Initialiser cudaOri
    if (hipSuccess != (errCuda = hipMemcpy(cudaOri, M, sizeM * sizeof(TYPE), hipMemcpyHostToDevice)))
       { printf("PB Copie Host ptDest -> cudaOri - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

    //Trier cudaOri par blocs de 2, puis par blocs de 4, etc jusqua 512
    for ( t = 1 ; t <= 512 and t<sizeM ; t= t*2){
        //partie divisible par d
        int d=t*2;
        int size_AetB = sizeM%d;//taille restante
        MergeSmallBatch_k<<<1024,1024>>>(cudaOri,sizeM-size_AetB,cudaDest,t*2);

        //partie restante
        int sizeA = min(size_AetB,t);
        int sizeB = size_AetB - sizeA;
        TYPE* cudaM = cudaDest+sizeM-size_AetB;
        TYPE* cudaA = cudaOri+sizeM-size_AetB;
        TYPE* cudaB = cudaA+sizeA;
        MergeSmall_k<<<1,sizeA+sizeB>>> (cudaA,cudaB,cudaM,sizeA,sizeB);

        // Flip Flop entre les bancs cudaDest et cudaOri
        TYPE * cudaTmp = cudaDest ;
        cudaDest = cudaOri ;
        cudaOri  = cudaTmp ;
    }

    //t=512 on trie par blocs de taille supérieure à laide de PathBig et MergeBig
    for ( t = t ; t < sizeM ; t= t*2)
    {
        for ( int i = 0 ; i < sizeM ; i = i + (2*t))
        {
            int sizeA = min(t,sizeM-i);
            int sizeB = min(t,max(sizeM-(i+t),0));

            TYPE * CudaVecteurA = cudaOri + i ;
            TYPE * CudaVecteurB = cudaOri + i + sizeA ;

            if ((sizeA == 0) || (sizeB == 0))
            {
                if (sizeA != 0)
                {
                  if (hipSuccess != (errCuda = hipMemcpy(cudaDest + i, CudaVecteurA , sizeA * sizeof(TYPE), hipMemcpyDeviceToDevice)))
                  { printf("PB Copie Cuda A -> ptDes rab %d - %d - %s \n",sizeA, errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }
                }
                continue ;
            }

            //Merge
            if (sizeA+sizeB <= 1024) { printf("Oups, on est censé avoir traité ce cas par le merge batch\n") ; return ;}
            int nbthread = 1024;
            int NbDiagonale  = (sizeA + sizeB) / nbthread ;
            if (NbDiagonale > 1024) { printf("Oups, on n'a pas fait le code pour nbDiag %d > 1024\n",NbDiagonale) ; return ; }
            int NbWindows    =  NbDiagonale ;
            NbWindows   += (((sizeA + sizeB) % nbthread) == 0)?0:1 ;  // si (SizeA + SizeB) % nbthread == 0 alors nbWindows = 0  sinon = 1


            PathBig<<<1,NbDiagonale>>>(CudaVecteurA, CudaVecteurB, sizeA , sizeB, CudaDiagBx, CudaDiagAy, nbthread,NbWindows) ;
            int nbBlock  = (sizeA+sizeB) / 1024 ;
            nbBlock += ((sizeA+sizeB) % 1024)?1:0 ;
            MergeBig_k<<<nbBlock,1024>>> (CudaVecteurA, CudaVecteurB, cudaDest+i, CudaDiagAy, CudaDiagBx, nbthread) ;
        } // End for i

        // Flip Flop entre les bancs cudaDest et cudaOri
        TYPE * cudaTmp = cudaDest ;
        cudaDest = cudaOri ;
        cudaOri  = cudaTmp ;

    }
    //cudaOri est entièrement trié

    //remettre dans M
    if (hipSuccess != hipMemcpy(M, cudaOri, sizeM * sizeof(TYPE), hipMemcpyDeviceToHost))
    { printf("PB copie cuda M -> host M \n") ; fflush(stdout);  exit(2) ; }


    // Free
    if (cudaOri != NULL) { hipFree(cudaOri) ; cudaOri = NULL ; }
    if (cudaDest != NULL) { hipFree(cudaDest) ; cudaDest = NULL ; }
    if (CudaDiagAy != NULL) { hipFree(CudaDiagAy) ; CudaDiagAy = NULL ; }
    if (CudaDiagBx != NULL) { hipFree(CudaDiagBx) ; CudaDiagBx = NULL ; }
}


int main(int argc, char ** argv)
{
    //déclaration
    int sizeM;
    TYPE* M;
    hipEvent_t Start; hipEvent_t Stop; hipEventCreate(&Start) ; hipEventCreate(&Stop) ;
    float temps_iter;
    srand(1998);
    for (int i=0; i<11; i++)
    {
        temps_iter=0;
        sizeM = (1024*pow(2,i));
        M = (TYPE *) malloc(sizeM * sizeof(TYPE));
        for (int i =0; i<sizeM; i++) M[i]=rand();

        hipEventRecord(Start);
        MergeSort(M,sizeM);
        hipEventRecord(Stop) ;

        check("tableau M après", sizeM, M);
        hipEventElapsedTime(&temps_iter, Start, Stop) ;

        free(M);
        printf("Duree pour %d : %f s\n",sizeM,temps_iter/1000) ;
    }

    return 0 ;
}
