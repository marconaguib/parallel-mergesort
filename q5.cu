#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BILLION  1000000000.0F

// Pour pouvoir experimenter les performances avec les différents types
// FMT  Permet d'avoir un % adapté pour le printf et donc de pas avoir de warning
#define TYPE int
#define FMT  "d"

typedef struct
{
   int x ;
   int y ;
}  Point ;


void Affiche (char * tabMsg, TYPE * ptBuffer, int NB)
{
   TYPE * pt = ptBuffer ;
   for ( int k = 0 ; k < NB  ; k++ , pt ++)
   {   printf(" - %s[%03d] = %6" FMT, tabMsg, k , *pt) ;
       if ((k % 5) == (4))
       {  printf("\n") ; fflush(stdout);  }

   }
   printf("\n") ;
  fflush(stdout);
}

int check(char * msg, int Nb, TYPE * pto)
{
    TYPE * pt1 = pto ;
    TYPE * pt2 = pto + 1 ;
    int i ;

    for (i = 0 ; i < Nb-1 ; i ++)
    {
        if (*pt1 > *pt2)
        { printf("Check %s pour %d - Erreur en position %d %" FMT " > %" FMT " \n", msg, Nb, i, *pt1, *pt2) ;
    //	return i ;
	exit(25) ;
        }
        pt1 ++ ; pt2 ++ ;
    }

    printf("Check %s pour %d est OK \n", msg, Nb) ;
    return 0 ;
}
__global__ void MergeSmallBatch_k(TYPE *M, int sizeM_tot, TYPE* N, int d)
{
    int i = threadIdx.x%d;
    int Qt = (threadIdx.x-i)/d;
    int gbx = Qt + blockIdx.x*(blockDim.x/d);
    if (threadIdx.x + blockIdx.x*blockDim.x >= sizeM_tot) return; //gerer les débordements

    int t = d/2;
    int sizeA = t;
    int sizeB = t;

    M=M+gbx*d;
    TYPE* A=M;
    TYPE* B=A+sizeA;


    Point K, P, Q;
    int offset ;

    if (i > sizeA)
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ;
         Q.x = K.x + offset ; Q.y = K.y - offset ;

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  N[i+gbx*d] = A[Q.y] ; }
                   else
                   {  N[i+gbx*d] = B[Q.x] ; }
                   break ;
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }
}
int main(int argc, char ** argv)
{
    //déclaration
    int N = 10000;
    hipError_t errCuda;
    TYPE* ABAB; //[A_0,B_0,A_1,B_1,...]
    TYPE* MM; // [M_0,M_1,...], les merges respectifs de [A_0,B_0,A_1,B_1,...]
    TYPE* cudaABAB;
    TYPE* cudaMM;


    for (int d=4; d<=1024; d=d*2)
    {
        float m1;
       hipEvent_t Start; hipEvent_t Stop; hipEventCreate(&Start) ; hipEventCreate(&Stop) ;


        int size_total=d*N;

        //allocation
        if ((ABAB = (TYPE *) malloc(size_total * sizeof(TYPE))) == NULL)
            { printf("PB allocation Vecteur Ori\n") ; exit (1) ; }
        if ((MM = (TYPE *) malloc(size_total * sizeof(TYPE))) == NULL)
            { printf("PB allocation Vecteur Dest\n") ; exit (1) ; }

        //initialisation
        srand(5);
        for (int i =0; i<size_total; i++)
        {
            if (i%(d/2)==0) ABAB[i] = rand()%100;
            else ABAB[i]=ABAB[i-1]+rand()%100;
        }
        // Il faut que tous les A et les B soient triés
        // Donc ABAB est trié par blocs de taille (d/2)


        //Allocation
        if (hipSuccess != (errCuda = hipMalloc((void**)&cudaABAB, size_total * sizeof(TYPE))))
            { printf("PB allocation CudaVecteurABAB - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }  // cleanup a rajouter pour plus propre
        if (hipSuccess != (errCuda = hipMalloc((void**)&cudaMM, size_total * sizeof(TYPE))))
            { printf("PB allocation CudaVecteurMM - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }  // cleanup a rajouter pour plus propre

        if (hipSuccess != (errCuda = hipMemcpy(cudaABAB, ABAB, size_total * sizeof(TYPE), hipMemcpyHostToDevice)))
        { printf("PB Copie ABAB -> cudaABAB - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }

        hipEventRecord(Start);
        MergeSmallBatch_k<<<1024,1024>>>(cudaABAB,size_total,cudaMM,d); //a revoir
        hipEventRecord(Stop);

        if (hipSuccess != hipMemcpy(MM, cudaMM, size_total * sizeof(TYPE), hipMemcpyDeviceToHost))
          { printf("PB copie cudaMM -> MM \n") ; fflush(stdout);  exit(2) ; }

        hipEventElapsedTime(&m1, Start, Stop) ;
        printf("Duree pour d = %4d : %f ms\n",d,m1) ;

        //free
        free(MM);
        free(ABAB);
        if (cudaABAB != NULL) { hipFree(cudaABAB) ; cudaABAB = NULL ; }
        if (cudaMM != NULL) { hipFree(cudaMM) ; cudaMM = NULL ; }

    }
    return 0 ;
}
