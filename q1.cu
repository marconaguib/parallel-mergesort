#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>

#define TYPE long
#define FMT  "ld"

// Taille des vecteurs
//define sizeA 500
//define sizeB 500

// Algo de remplissage
enum EnumTypeRemplissage { AlgoPairImpair , AlgoRandom } ;
char MSG[sizeof(EnumTypeRemplissage)][64] =
    { "Algo A Pair et B Impair" ,
      "Algo remplissage aleatoire"
    } ;

EnumTypeRemplissage typeRemplissage = AlgoRandom ;

// Algo de Merge : Différents algorithmes pour pour faire le merge
enum EnumMerge {   AlgoMergePathGPU , AlgoMergeSmall_k, AlgoMergeSmallShared_k } ;

char MSGMerge[sizeof(EnumMerge)][64] =
   {  "Merge via MergePathGPU" ,
     "Merge via PathGPU en // pour les threads en parallèle", "Merge via PathGPU en // pour la mémoire shared"
   } ;

EnumMerge Algo = AlgoMergeSmallShared_k ;

// On parle de la notion de point d
typedef struct
{
   int x ;
   int y ;
}  Point ;

// Algorithme B en gpu
__global__ void MergePathGPU(TYPE *A, TYPE *B, TYPE *M, int cardA, int cardB)
{
    Point K;
    Point P;
    Point Q;
    int offset ;
    // int i = threadIdx.x ; // Id du thread, permet de savoir quelle valeur va être rangé à sa place définitive.
    for (int i = 0 ; i < cardA + cardB ;  i ++)
    {
    	if (i > cardA)
      {
    		K.x = i - cardA ; K.y = cardA ;
    		P.x = cardA ; P.y = i - cardA ;
    	}
    	else
      {
    		K.x = 0 ; K.y = i ;
    		P.x = i ; P.y = 0 ;
    	}
    	while (1)
      {
        offset = abs (K.y - P.y) / 2 ;
        Q.x = K.x + offset ; Q.y = K.y - offset ; // Q est bien sur une diagonale à 45°

    		if (((Q.y >= 0 ) && (Q.x <= cardB)) &&
    			((Q.y == cardA) || (Q.x == 0) || (A[Q.y]>B[Q.x -1]))){

    			if ((Q.x == cardB) || (Q.y == 0) || (A[Q.y-1]<=B[Q.x]))
          {
    				if((Q.y < cardA) && ((Q.x == cardB) || (A[Q.y] <= B[Q.x])))
            {  M[i]= A[Q.y] ; }
    				else
            {	 M[i] = B[Q.x] ; }
    				break;  // Pour simuler passage au thread suivant
    			}
    			else
          {  K.x = Q.x +1 ; K.y = Q.y - 1 ; }
    		}
    		else
        {	P.x = Q.x -1 ; P.y = Q.y +1 ; }
    	}
    }
} // End of MergePathGPU

__global__ void MergeSmall_k(TYPE *A, TYPE *B, TYPE *M, int cardA, int cardB)
{
    Point K;
    Point P;
    Point Q;
    int offset ;
    int i = threadIdx.x ; // Id du thread, permet de savoir quelle valeur va être rangé à sa place définitive.
    // for (int i = 0 ; i < cardA + cardB ;  i ++)
    {
    	if (i > cardA)
      {
    		K.x = i - cardA ; K.y = cardA ;
    		P.x = cardA ; P.y = i - cardA ;
    	}
    	else
      {
    		K.x = 0 ; K.y = i ;
    		P.x = i ; P.y = 0 ;
    	}
    	while (1)
      {
        offset = abs (K.y - P.y) / 2 ;
        Q.x = K.x + offset ; Q.y = K.y - offset ; // Q est bien sur une diagonale à 45°

    		if (((Q.y >= 0 ) && (Q.x <= cardB)) &&
    			((Q.y == cardA) || (Q.x == 0) || (A[Q.y]>B[Q.x -1]))){

    			if ((Q.x == cardB) || (Q.y == 0) || (A[Q.y-1]<=B[Q.x]))
          {
    				if((Q.y < cardA) && ((Q.x == cardB) || (A[Q.y] <= B[Q.x])))
            {  M[i]= A[Q.y] ; }
    				else
            {	 M[i] = B[Q.x] ; }
    				break;  // Pour simuler passage au thread suivant
    			}
    			else
          {  K.x = Q.x +1 ; K.y = Q.y - 1 ; }
    		}
    		else
        {	P.x = Q.x -1 ; P.y = Q.y +1 ; }
    	}
    }
} // End of MergeSmall_k

__global__ void MergeSmallShared_k(TYPE *GlobalCudaA, TYPE *GlobalCudaB, TYPE *M, int sizeA, int sizeB)
{
    extern __shared__ TYPE dataAB[] ;
    unsigned int tid = threadIdx.x;
    unsigned int i   = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements

 // Chargement des données dans la mémoire partagée par le thread ;
     dataAB[tid] = (i < sizeA)?GlobalCudaA[i]:GlobalCudaB[i-sizeA] ;

 // On attend qur tous les threads aient faits le travail
    __syncthreads();

 // On recadre nos pointeurs pourqu'ils pointent vers la mémoire partagée et la globale
    TYPE * A = dataAB ;
    TYPE * B = dataAB + sizeA ;

    Point K, P, Q;
    int offset ;

    if (i > sizeA)
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ;
         Q.x = K.x + offset ; Q.y = K.y - offset ;

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ;
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

} // End of MergeSmallShared_k

// Pour le debug, On peut afficher le tableau
void Affiche(char * tabMsg, TYPE * ptBuffer, int NB)
{
   TYPE * pt = ptBuffer ;
   for ( int k = 0 ; k < NB  ; k++ , pt ++)
   {   printf(" - %s[%03d] = %6" FMT, tabMsg, k , *pt) ;
       if ((k % 5) == (4))
       {  printf("\n") ; }
   }
   printf("\n") ;
}

// Pour vérifier que le tableau est trié
int check(char * msg, int Nb, TYPE * pto)
{
    TYPE * pt1 = pto ;
    TYPE * pt2 = pto + 1 ;

    for (int i = 0 ; i < Nb-1 ; i ++)
    {
        if (*pt1 > *pt2)
        { printf("Check %s pour %d - Erreur en position %d %" FMT " > %" FMT " \n", msg, Nb, i, *pt1, *pt2) ;
          return i ;
        }
        pt1 ++ ; pt2 ++ ;
    }

    printf("Check %s pour %d est OK \n", msg, Nb) ;
    return 0 ;
}

// Compare 2 tableaux résultats (en fait l égalité entre 2 tableaux)
void compareResult(TYPE * res1, TYPE * res2, int Nb)
{
    int nberr = 0 ;

    for (int i = 0 ; i < Nb ; i ++, res1 ++, res2 ++)
    {
        if (*res1 != *res2)
        {   nberr ++ ;
            if (nberr < 16) // Seuil d affichage
            {
                printf("En position %d, %" FMT " != %" FMT "\n",i,*res1,*res2) ;
            }
        }
    }

    if (nberr == 0)
    { printf("Les 2 vecteurs de %d elements sont identiques.\n",Nb) ; }
    else
    { printf("Les 2 vecteurs differents en %d points (sur %d)\n",nberr, Nb) ; }
}

// Déclaration
   TYPE * HostVecteurA = NULL ;
   TYPE * HostVecteurB = NULL ;
   TYPE * HostVecteurC = NULL ;
   TYPE * HostVecteurD = NULL ;

   TYPE * CudaVecteurA = NULL ;
   TYPE * CudaVecteurB = NULL ;
   TYPE * CudaVecteurC = NULL ;

void cleanup()
{
    if (HostVecteurA != NULL) { free(HostVecteurA) ; }
    if (HostVecteurB != NULL) { free(HostVecteurB) ; }
    if (HostVecteurC != NULL) { free(HostVecteurC) ; }
    if (HostVecteurD != NULL) { free(HostVecteurD) ; }

    if (CudaVecteurA != NULL) { hipFree(CudaVecteurA) ; }
    if (CudaVecteurB != NULL) { hipFree(CudaVecteurB) ; }
    if (CudaVecteurC != NULL) { hipFree(CudaVecteurC) ; }
}

int mesure(int sizeA , int sizeB , EnumMerge Algo)
{

// Hello
   printf("CardA %d - CardB %d - Algo de remplissage %s - %s\n",
          sizeA, sizeB, MSG[typeRemplissage],MSGMerge[Algo]) ;

// allocation dynamique Sur le host
   if (( HostVecteurA = (TYPE *) malloc(sizeA * sizeof(TYPE))) == NULL)
   { printf("PB allocation HostVecteurA\n") ; cleanup() ; exit(1) ; }

   if ((HostVecteurB = (TYPE *) malloc(sizeB * sizeof(TYPE))) == NULL)
   { printf("PB allocation HostVecteurB\n") ; cleanup() ; exit(1) ; }

   if ((HostVecteurC = (TYPE *) malloc((sizeA + sizeB) * sizeof(TYPE))) == NULL)
   { printf("PB allocation HostVecteurC\n") ; cleanup() ; exit(1) ; }

   if ((HostVecteurD = (TYPE *) malloc((sizeA + sizeB) * sizeof(TYPE))) == NULL)
   { printf("PB allocation HostVecteurD\n") ; cleanup() ; exit(1) ; }

// Initialisation des deux vecteurs de base sur le Host
   switch (typeRemplissage)
   { case AlgoPairImpair:
       for (int j = 0 ; j < sizeA; j ++) { HostVecteurA[j] = 2 * j ; }
       for (int j = 0 ; j < sizeB; j ++) { HostVecteurB[j] = 2 * j + 1 ; }
       break ;
     case AlgoRandom:
       int N = 100 ;
       // srand (time (NULL));
       srand (1925) ;
       HostVecteurA[0] = rand() % N ;
       HostVecteurB[0] = rand() % N ;
       for (int j = 1 ; j < sizeA ; j ++)
          { HostVecteurA[j] = HostVecteurA[j-1] + (rand() % N) ; }
       for (int j = 1 ; j < sizeB ; j ++)
          { HostVecteurB[j] = HostVecteurB[j-1] + (rand() % N) ; }
       break ;
   }

 // Vérifier qu on génére bien les tableaux
    check("Vecteur A ", sizeA, HostVecteurA) ;
    check("Vecteur B ", sizeB, HostVecteurB) ;

 // Mesures de temps
    hipEvent_t Start, StartAllocA, StartAllocB, StartAllocC, StartPushA, StartPushB, StartGetC, StartMerge ;
    hipEvent_t Stop,  StopAllocA, StopAllocB, StopAllocC, StopPushA, StopPushB, StopGetC, StopMerge ;

    hipEventCreate(&Start) ; hipEventCreate(&StartAllocA) ; hipEventCreate(&StartAllocB) ; hipEventCreate(&StartAllocC) ;
    hipEventCreate(&StartPushA) ; hipEventCreate(&StartPushB) ; hipEventCreate(&StartGetC) ; hipEventCreate(&StartMerge) ;

    hipEventCreate(&Stop) ; hipEventCreate(&StopAllocA) ; hipEventCreate(&StopAllocB) ; hipEventCreate(&StopAllocC) ;
    hipEventCreate(&StopPushA) ; hipEventCreate(&StopPushB) ; hipEventCreate(&StopGetC) ; hipEventCreate(&StopMerge) ;

 // Allocation dynamique sur le GPU
    hipEventRecord(Start);
    hipEventRecord(StartAllocA);
    if (hipSuccess != hipMalloc((void**)&CudaVecteurA, sizeA * sizeof(TYPE)))
    { printf("PB allocation CudaVecteurA\n") ; cleanup() ; exit (1) ; }
    hipEventRecord(StopAllocA); hipEventSynchronize(StopAllocA) ;

    hipEventRecord(StartAllocB);
    if (hipSuccess != hipMalloc((void**)&CudaVecteurB, sizeB * sizeof(TYPE)))
    { printf("PB allocation CudaVecteurB\n") ; cleanup() ; exit (1) ; }
    hipEventRecord(StopAllocB); hipEventSynchronize(StopAllocB) ;

    hipEventRecord(StartAllocC);
    if (hipSuccess != hipMalloc((void**) &CudaVecteurC, (sizeA + sizeB) * sizeof(TYPE)))
    { printf("PB allocation CudaVecteurC\n") ; cleanup() ; exit (1) ; }
    hipEventRecord(StopAllocC); hipEventSynchronize(StopAllocC) ;

// Recopie Host => GPU
   hipEventRecord(StartPushA);
   if (hipSuccess != hipMemcpy(CudaVecteurA, HostVecteurA,sizeA * sizeof(TYPE), hipMemcpyHostToDevice))
   { printf("PB copie Hosta -> cuda A\n") ; cleanup() ; exit(2) ; }
   hipEventRecord(StopPushA); hipEventSynchronize(StopPushA) ;

   hipEventRecord(StartPushB);
   if (hipSuccess != hipMemcpy(CudaVecteurB, HostVecteurB,sizeB * sizeof(TYPE), hipMemcpyHostToDevice))
   { printf("PB copie Hosta -> cuda B\n") ; cleanup() ; exit(2) ; }
   hipEventRecord(StopPushB); hipEventSynchronize(StopPushB) ;

   hipEventRecord(StartMerge) ;
   switch(Algo)
   {
       case AlgoMergePathGPU:
            MergePathGPU<<<1,1>>>(CudaVecteurA, CudaVecteurB, CudaVecteurC, sizeA, sizeB) ;
       break ;
       case AlgoMergeSmall_k:
            if (sizeA + sizeB > 1024)
            {  printf("Cet algo ne fonctionne que pour sizeA %d + sizeB %d < 1024 (%d)\n",
                     sizeA, sizeB, (sizeA +sizeB)) ;
            }
            else //
            {  MergeSmall_k<<<1,sizeA+sizeB>>>(CudaVecteurA, CudaVecteurB, CudaVecteurC, sizeA, sizeB) ; }
        case AlgoMergeSmallShared_k:
            if (sizeA + sizeB > 1024)
            {  printf("Cet algo ne fonctionne que pour sizeA %d + sizeB %d < 1024 (%d)\n",
                     sizeA, sizeB, (sizeA +sizeB)) ;
            }
            else // 3eme argument permet de faire la réservation de la mémoire __shared__
            {  MergeSmallShared_k<<<1,sizeA+sizeB,(sizeA+sizeB) * sizeof(TYPE)>>>(CudaVecteurA, CudaVecteurB, CudaVecteurC, sizeA, sizeB) ; }

       break ;
   }

   hipEventRecord(StopMerge) ; hipEventSynchronize(StopMerge) ;

// On recupere le resultat donc GPU => CPU
   hipEventRecord(StartGetC) ;
   if (hipSuccess != hipMemcpy(HostVecteurC, CudaVecteurC, (sizeA + sizeB) * sizeof(TYPE), hipMemcpyDeviceToHost))
   { printf("PB copie cuda C -> host C \n") ; cleanup() ; exit(2) ; }
   hipEventRecord(StopGetC) ; hipEventSynchronize(StopGetC) ;

   hipEventRecord(Stop) ; hipEventSynchronize(Stop) ;

// --- Affichage des temps de traitement ---
   float m1;
   //float  m2, m3 ;
   //hipEventElapsedTime(&m1, StartAllocA, StopAllocA) ;
   //hipEventElapsedTime(&m2, StartAllocB, StopAllocB) ;
   //hipEventElapsedTime(&m3, StartAllocC, StopAllocC) ;

  /* printf("Allocation A (sizeA %d) %f ms - B (sizeB %d) %f ms - C (sizeC %d) %f ms \n",
           sizeA, m1, sizeB , m2 , sizeA + sizeB , m3) ;

   hipEventElapsedTime(&m1, StartPushA, StopPushA) ;
   hipEventElapsedTime(&m2, StartPushB, StopPushB) ;
   hipEventElapsedTime(&m3, StartGetC,  StopGetC) ;

   printf("PushA (sizeA %d) %f ms - Débit %f Mo/s - PushB (sizeB %d) %f ms - Débit %f Mo/s \n",
           sizeA*sizeof(TYPE), m1, (sizeof(TYPE) * sizeA) / m1 / 1000 ,
           sizeB*sizeof(TYPE), m2, (sizeof(TYPE) * sizeB) / m2 / 1000 ) ;

   printf("GetC (sizeC %d) %f ms - Débit %f Mo/s\n",
           (sizeA+sizeB)*sizeof(TYPE), m3, (sizeof(TYPE) * (sizeA + sizeB)) / m3 / 1000 ) ;
 */

   hipEventElapsedTime(&m1, StartMerge, StopMerge) ;
   printf("Duree %f ms\n",m1) ;
    // hipEvent_t Start, StartAllocA, StartAllocB, StartAllocC, StartPushA, StartPushB, StartGetC, StartMerge ;
    // hipEvent_t Stop,  StopAllocA, StopAllocB, StopAllocC, StopPushA, StopPushB, StopGetC, StopMerge ;

// --- Un peu de travail sur le HOST, pour valider nos résultats ---
// Le vecteur résultat est trié, mais pas nécessairement juste
   check("Vecteur M ",sizeA+sizeB, HostVecteurC) ;



   // Affiche("VectC", HostVecteurC, sizeA + sizeB) ;

   cleanup() ; printf("Fin de mesure \n") ;
   return 0 ;
}

int main(int argc , char ** argv)
{

    mesure( 50,  50, AlgoMergePathGPU) ;
    mesure(100, 100, AlgoMergePathGPU) ;
    mesure(150, 150, AlgoMergePathGPU) ;
    mesure(200, 200, AlgoMergePathGPU) ;
    mesure(250, 250, AlgoMergePathGPU) ;
    mesure(300, 300, AlgoMergePathGPU) ;
    mesure(350, 350, AlgoMergePathGPU) ;
    mesure(400, 400, AlgoMergePathGPU) ;
    mesure(450, 450, AlgoMergePathGPU) ;
    mesure(500, 500, AlgoMergePathGPU) ;

    mesure( 50,  50, AlgoMergeSmall_k) ;
    mesure(100, 100, AlgoMergeSmall_k) ;
    mesure(150, 150, AlgoMergeSmall_k) ;
    mesure(200, 200, AlgoMergeSmall_k) ;
    mesure(250, 250, AlgoMergeSmall_k) ;
    mesure(300, 300, AlgoMergeSmall_k) ;
    mesure(350, 350, AlgoMergeSmall_k) ;
    mesure(400, 400, AlgoMergeSmall_k) ;
    mesure(450, 450, AlgoMergeSmall_k) ;
    mesure(500, 500, AlgoMergeSmall_k) ;

    mesure( 50,  50, AlgoMergeSmallShared_k) ;
    mesure(100, 100, AlgoMergeSmallShared_k) ;
    mesure(150, 150, AlgoMergeSmallShared_k) ;
    mesure(200, 200, AlgoMergeSmallShared_k) ;
    mesure(250, 250, AlgoMergeSmallShared_k) ;
    mesure(300, 300, AlgoMergeSmallShared_k) ;
    mesure(350, 350, AlgoMergeSmallShared_k) ;
    mesure(400, 400, AlgoMergeSmallShared_k) ;
    mesure(450, 450, AlgoMergeSmallShared_k) ;
    mesure(500, 500, AlgoMergeSmallShared_k) ;

    return 0 ;
}
