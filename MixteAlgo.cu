#include "MyInc.h"

// Pour la question 2 
void MergeWindowsGPU(TYPE * CudaVecteurA, TYPE * CudaVecteurB, TYPE * CudaVecteurC, int SizeA, int SizeB, int * CudaDiagAy , int * CudaDiagBx , int * HostDiagAy, int * HostDiagBx ,  
                     TYPE * HostVecteurA , TYPE * HostVecteurB, TYPE * HostVecteurC , int nbthread, int NbDiagonale, int NbWindows)
{

   hipError_t errCuda ;

   if (1024 < NbDiagonale) 
   {  printf("Mode TriWindowsGPU_para code réduit pour NBDiag < 1024\n") ; return ; }  

/* Le caclul des diagonales peut se faire soit en local soit en CGPU. 
   Quelque soit le lieu de calcul, il sera néceassaire de recopuer les vecteur A et B sur la carte GPU. 
   Si la cul est local, il est nécessaire d'en faire la copie vers la carte GPU. 
   Pour un petit nombre de diagonale, le celcul peut être intéressant, s'il est plus court que l'initialiasitation
   des cacluls sur la carte. 
*/

   if (0) // Pour la construction par étape
   {
       HostDiagBx[0] = HostDiagAy[0] = 0 ; // Top en haut à gauche
       HostDiagBx[NbWindows] = SizeB ; HostDiagAy[NbWindows] = SizeA ; // Coin n bas à droite

    // Calcul de la position basse des fenêtres
       for (int i = 0 ; i < NbDiagonale ; i ++) // Simul le // 
       {
           AnalyseDiagonales(HostVecteurA, HostVecteurB, SizeA , SizeB, HostDiagBx, HostDiagAy, nbthread, i)   ;
       } 

    // On recopie notre vecteur diagonale vers le device
       if (hipSuccess != (errCuda = hipMemcpy(CudaDiagBx, HostDiagBx, (NbWindows+1) * sizeof(int), hipMemcpyHostToDevice)))
       { printf("PB copie DiagBx -> cuda - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; cleanup() ; exit(2) ; }
 
       if (hipSuccess != (errCuda = hipMemcpy(CudaDiagAy, HostDiagAy, (NbWindows+1) * sizeof(int), hipMemcpyHostToDevice)))
       { printf("PB copie DiagAy -> cuda - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; cleanup() ; exit(2) ; }
 
       // AfficheDiag(HostDiagAy,HostDiagBx,NbWindows) ; 

   }
   else
   {
      // Il faut initialisaer le vecteur des diagonales - Soit on fait une copie full, soit on fait une oppie en debut et en fin de veteur, soit on fait un appel à 
      // une fonction sur le GPU 
         initDiagGPU<<<1,1>>>(SizeA , SizeB, CudaDiagBx, CudaDiagAy, NbWindows)   ;
      // Pour l'instant on ne sait faire que nbDiagonale < 1024 (pas de gestion de bloc)A
         AnalyseDiagonalesGPU<<<1,NbDiagonale>>>(CudaVecteurA, CudaVecteurB, SizeA , SizeB, CudaDiagBx, CudaDiagAy, nbthread) ; 

      // Dans le cas où nous aurions besoin des coordonnées de la diagonale en local, il faudrait refaire la copie 
       if (hipSuccess != (errCuda = hipMemcpy(HostDiagBx, CudaDiagBx, (NbWindows+1) * sizeof(int), hipMemcpyDeviceToHost)))
       { printf("PB copie cuda -> DiagBx - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; cleanup() ; exit(2) ; }
 
       if (hipSuccess != (errCuda = hipMemcpy(HostDiagAy, CudaDiagAy, (NbWindows+1) * sizeof(int), hipMemcpyDeviceToHost)))
       { printf("PB copie cuda -> DiagAy - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; cleanup() ; exit(2) ; }

//       AfficheDiag(HostDiagAy,HostDiagBx,NbWindows) ; 
 
   }  

   if ((SizeA + SizeB) < 1024) 
   {   mergeGPU<<<1,SizeA+SizeB>>> (CudaVecteurA, CudaVecteurB, CudaVecteurC, CudaDiagAy, CudaDiagBx, nbthread) ; } 
   else // Il faut decouper en bloc 
   { // Traille d'un bloc et nb bloc  
        int nbBlock  = (SizeA+SizeB) / 1024 ; 
        nbBlock += ((SizeA+SizeB) % 1024)?1:0 ; 
        mergeGPU<<<nbBlock,1024>>> (CudaVecteurA, CudaVecteurB, CudaVecteurC, CudaDiagAy, CudaDiagBx, nbthread) ; 
   } 

} 
