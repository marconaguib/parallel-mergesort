#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "MyInc.h" 

// Algorithme A
void MergeSimpleHOST(TYPE *A, TYPE *B, TYPE *M, int sizeA, int sizeB)
{
    int j = 0;
    int i = 0;

    // On utilise comme pointeurs les arguments de la fonctions
    while (i + j < sizeA + sizeB )
    {
        if (i >= sizeA ) // On a épuisé tout les elts de A, donc on complete avec B
        {  *M = *B; // on utilise les pointeurs pour éviter de faire l opération i+j et se déplacer  = gain de performance
            M = M + 1 ; // Je déplace les pointeurs
            B = B + 1 ;
            j = j + 1 ;
        }
        else if ((j >= sizeB) || (*A < *B))
        {   *M = *A ; M = M + 1 ; A = A + 1 ; i = i + 1 ; }
        else
        {   *M = *B ; M = M + 1 ; B = B + 1 ; j = j + 1 ; }
    }
} // End of MergeSimpleHOST

// Algorithme B
void MergePathHOST(TYPE *A, TYPE *B, TYPE *M, int sizeA, int sizeB)
{
    Point K;
    Point P;
    Point Q;

    int i ; int offset ; 
    for (i = 0 ; i < sizeA + sizeB ; i ++)
    {
	   if (i > sizeA) 
        {
    		K.x = i - sizeA ; K.y = sizeA ;
    		P.x = sizeA ; P.y = i - sizeA ;
    	  }
    	else // x ~ horizontal
        {
    		K.x = 0 ; K.y = i ;
    		P.x = i ; P.y = 0 ;
    	  }
    	while (1)
        {
             offset = abs(K.y - P.y) / 2 ; 
             Q.x = K.x + offset ; Q.y = K.y - offset ; 

             if ( (Q.y >= 0) && (Q.x <= sizeB) &&
                  ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
             {
                  if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
                  {
                       if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                       {  M[i] = A[Q.y] ; }
                       else
                       {  M[i] = B[Q.x] ; }
                       break ; // Pour simuler passage au thread suivant
                  }
                  else
                  {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
             }
             else
             { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    	}
    }
} // End of MergePathHOST

// Travail  préparation, afin d'avoir un code parallélsable sur GPU
void AnalyseDiagonales(TYPE * A, TYPE * B, int sizeA, int sizeB, int * DiagBx, int * DiagAy, int nbthread, int nth) 
{
    Point K, P, Q ; 
    int   px , py ; 
    TYPE  v;  // Génère un warning sur les ppti-gpu* 
 
    int offset ; 
    int numDiag  = (nth+1) * nbthread -1 ; // Les tableaux vont de 0 à N-1 
	if (numDiag > sizeA) 
        {
    		K.x = numDiag - sizeA ; K.y = sizeA ;
    		P.x = sizeA ; P.y = numDiag - sizeA ;
    	}
    	else // x ~ horizontal
        {
    		K.x = 0 ; K.y = numDiag ;
    		P.x = numDiag ; P.y = 0 ;
    	}
    	while (1)
        {
             offset = abs(K.y - P.y) / 2 ; 
             Q.x = K.x + offset ; Q.y = K.y - offset ; 

             if ( (Q.y >= 0) && (Q.x <= sizeB) &&
                  ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
             {
                  if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
                  {
                       px = Q.x ; py = Q.y ;
                       if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                       {  v = A[Q.y] ; 
                          py ++ ; 
                       }
                       else
                       {  v = B[Q.x] ; 
                          px ++ ; 
                       }
                       // printf("Analyse Diagonale Point de Sortie ref %d - M %" FMT " Q (A Q.y %d) (B Q.x %d) rv.x %d rv.y %d\n",i,v,Q.y,Q.x,rv->x,rv->y) ; 
                       DiagBx[nth+1] = px ; DiagAy[nth+1] = py ; 
                       break ; // Pour simuler passage au thread suivant
                  }
                  else
                  {  K.x = Q.x + 1 ; K.y = Q.y - 1 ;  }
             }
             else
             { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    	}
} // End of AnalyseDiagonales

