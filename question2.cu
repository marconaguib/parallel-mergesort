// Version single file 

#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <stdio.h>

// Pour pouvoir experimenter les performances avec les différents types
// FMT  Permet d'avoir un % adapté pour le printf et donc de pas avoir de warning
#define TYPE int 
#define FMT  "d"

typedef struct 
{
   int x ; 
   int y ; 
}  Point ; 


__global__ void PathBig(TYPE * CudaVecteurA, TYPE * CudaVecteurB, int sizeA , int sizeB, int * CudaDiagBx, int * CudaDiagAy, int nbthread, int NbWindows)  
{
    //Initialisation diagolane 
    CudaDiagBx[0] = CudaDiagAy[0] = 0 ; 
    CudaDiagBx[NbWindows] = sizeB ;  
    CudaDiagAy[NbWindows] = sizeA ;  
   
    int nth = threadIdx.x; // On explore le nth diagonale 
    Point K, P, Q ; 
    int   px , py ; 
    TYPE * A = CudaVecteurA ; 
    TYPE * B = CudaVecteurB ; 
    int offset ; 
    int numDiag  = (nth+1) * nbthread -1 ; // Les tableaux vont de 0 à N-1 
	  if (numDiag > sizeA) 
        {
    		K.x = numDiag - sizeA ; K.y = sizeA ;
    		P.x = sizeA ; P.y = numDiag - sizeA ;
    	}
    	else // x ~ horizontal
        {
    		K.x = 0 ; K.y = numDiag ;
    		P.x = numDiag ; P.y = 0 ;
    	}
    	while (1)
        {
             offset = abs(K.y - P.y) / 2 ; 
             Q.x = K.x + offset ; Q.y = K.y - offset ; 

             if ( (Q.y >= 0) && (Q.x <= sizeB) &&
                  ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
             {
                  if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
                  {
                       px = Q.x ; py = Q.y ;
                       if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                       {  // v = A[Q.y] ; 
                          py ++ ; 
                       }
                       else
                       {  // v = B[Q.x] ; 
                          px ++ ; 
                       }
                       // printf("Analyse Diagonale Point de Sortie ref %d - M %" FMT " Q (A Q.y %d) (B Q.x %d) rv.x %d rv.y %d\n",i,v,Q.y,Q.x,rv->x,rv->y) ; 
                       CudaDiagBx[nth+1] = px ; CudaDiagAy[nth+1] = py ; 
                       break ; // Pour simuler passage au thread suivant
                  }
                  else
                  {  K.x = Q.x + 1 ; K.y = Q.y - 1 ;  }
             }
             else
             { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    	}
} // End of PathBig

__global__ void MergeBig_k(TYPE * CudaVecteurA, TYPE * CudaVecteurB, TYPE * CudaVecteurC, int * CudaDiagAy, int * CudaDiagBx , int nbthread) 
{ 	

    // int i = threadIdx.x ;     // On renge le Ieme element
    int i = blockIdx.x * blockDim.x + threadIdx.x; // On range le ieme elet 
    int diag = (i / nbthread)  ;   // Dans quel fenêtre est-il ?  
    int indC = nbthread * diag ; 
    
    TYPE *A = CudaVecteurA+CudaDiagAy[diag] ; 
    TYPE *B = CudaVecteurB+CudaDiagBx[diag] ; 
    TYPE *M = CudaVecteurC + indC  ;  
    int sizeA = CudaDiagAy[diag+1]-CudaDiagAy[diag] ; 
    int sizeB = CudaDiagBx[diag+1]-CudaDiagBx[diag] ; 

    Point K, P, Q;
    int offset ;
 
    i = i % nbthread ; // On recadre i dans le nouvel espace
    if (i >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements
    if (i > sizeA) 
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ; 
         Q.x = K.x + offset ; Q.y = K.y - offset ; 

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ; 
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

}

int check(char * msg, int Nb, TYPE * pto)
{
    TYPE * pt1 = pto ;
    TYPE * pt2 = pto + 1 ;
    int i ;

    for (i = 0 ; i < Nb-1 ; i ++)
    {
        if (*pt1 > *pt2)
        { printf("Check %s pour %d - Erreur en position %d %" FMT " > %" FMT " \n", msg, Nb, i, *pt1, *pt2) ;
          return i ;
        }
        pt1 ++ ; pt2 ++ ;
    }
    printf("Premiere valeur de %s = %" FMT ", deuxième valeur = %" FMT ", troisième valeur = %" FMT " \n", msg, pto[0], pto[1], pto[2]);
    printf("Check %s pour %d est OK \n", msg, Nb) ;
    return 0 ;
}

void Affiche(char * tabMsg, TYPE * ptBuffer, int NB)
{  
   TYPE * pt = ptBuffer ; 
   for ( int k = 0 ; k < NB  ; k++ , pt ++) 
   {   printf(" - %s[%03d] = %6" FMT, tabMsg, k , *pt) ; 
       if ((k % 5) == (4)) 
       {  printf("\n") ; }
   }
   printf("\n") ;
}

int main(int argc, char ** argv) 
{
    //déclaration
    int sizeA = 1600;
    int sizeB = 1000 ;
    int sizeM = sizeA + sizeB ; 
    TYPE* A; 
    TYPE* B ; 
    TYPE* M;
 
    float m1; 
    hipEvent_t Start; hipEvent_t Stop; hipEventCreate(&Start) ; hipEventCreate(&Stop) ; 
 
    //allocation
    if ((A = (TYPE *) malloc(sizeA * sizeof(TYPE))) == NULL)
        { printf("PB allocation VecteurA\n") ; exit (1) ; }
 
    if ((B= (TYPE *) malloc(sizeB * sizeof(TYPE))) == NULL)
        { printf("PB allocation VecteurB\n") ; exit (1) ; }
 
    if ((M= (TYPE *) malloc(sizeM * sizeof(TYPE))) == NULL)
        { printf("PB allocation VecteurM\n") ; exit (1) ; }
 
    //initialisation
    srand(1925);
    A[0] = B[0] = rand()%100;
    for (int i =1; i<sizeA; i++)
    {
        A[i]=A[i-1]+rand()%100;
    }
    for (int i =1; i<sizeB; i++)
    {
        B[i]=B[i-1]+rand()%100;
    }
 
  //Declarations
    hipError_t errCuda;
    TYPE * CudaVecteurA = NULL ; 
    TYPE * CudaVecteurB = NULL ; 
    TYPE * CudaVecteurM = NULL ; 

    int nbthread = 512;  // a verifier 
    int NbDiagonale  = (sizeA + sizeB) / nbthread ;
    int NbWindows    = NbDiagonale ; 
    NbWindows       += (((sizeA + sizeB) % nbthread) == 0)?0:1 ;  // si (SizeA + SizeB) % nbthread == 0 alors nbWindows = 0  sinon = 1
    int  * CudaDiagBx   = NULL ;
    int  * CudaDiagAy   = NULL ;
 
    //Allocation 
    if (hipSuccess != (errCuda = hipMalloc((void**)&CudaVecteurA, sizeA * sizeof(TYPE))))
        { printf("PB allocation CudaVecteurA - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout) ; exit (1) ; } // cleanup a rajouter pour plus propre
    if (hipSuccess != (errCuda = hipMalloc((void**)&CudaVecteurB, sizeB * sizeof(TYPE))))
        { printf("PB allocation CudaVecteurB - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }  // cleanup a rajouter pour plus propre
    if (hipSuccess != (errCuda = hipMalloc((void**)&CudaVecteurM, sizeM * sizeof(TYPE))))
        { printf("PB allocation CudaVecteurM - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; } 

     if (hipSuccess != (errCuda = hipMalloc((void**)&CudaDiagBx, (NbWindows + 1) * sizeof(int))))
        { printf("PB allocation CudaDiagBx - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout); exit (1) ; }
   
     if (hipSuccess != (errCuda = hipMalloc((void**)&CudaDiagAy, (NbWindows + 1)* sizeof(int))))
       { printf("PB allocation CudaDiagAy - %d - %s \n",errCuda,hipGetErrorName(errCuda)) ; fflush(stdout);  exit (1) ; }
    
    if (hipSuccess != hipMemcpy(CudaVecteurA, A, sizeA * sizeof(TYPE), hipMemcpyHostToDevice))
        { printf("PB copie host A -> cuda A \n") ; fflush(stdout);  exit(2) ; } 
    
    if (hipSuccess != hipMemcpy(CudaVecteurB, B, sizeB * sizeof(TYPE), hipMemcpyHostToDevice))
        { printf("PB copie host B -> cuda B \n") ; fflush(stdout);  exit(2) ; } 
 
    hipEventRecord(Start);

    PathBig<<<1,NbDiagonale>>>(CudaVecteurA, CudaVecteurB, sizeA , sizeB, CudaDiagBx, CudaDiagAy, nbthread,NbWindows) ;
 
    int nbBlock  = (sizeA+sizeB) / 1024 ; 
    nbBlock += ((sizeA+sizeB) % 1024)?1:0 ; 
 
 if (sizeM <1024)
 {
     printf("La fonction MergeBig ne peut pas être prise en compte car sizeA+sizeB <1024");
    exit(2);
 }

  else { MergeBig_k<<<nbBlock,1024>>> (CudaVecteurA, CudaVecteurB, CudaVecteurM, CudaDiagAy, CudaDiagBx, nbthread) ;}

  if (hipSuccess != hipMemcpy(M, CudaVecteurM, sizeM * sizeof(TYPE), hipMemcpyDeviceToHost))
        { printf("PB copie cuda M -> host M \n") ; fflush(stdout);  exit(2) ; }
    hipEventRecord(Stop) ; 


    check((char *)"Check tableau M après", sizeM, M);
    hipEventElapsedTime(&m1, Start, Stop) ; 
    printf("Duree %f s\n",m1/1000) ;
    //Affiche ("Tableau M", M, sizeM); 
  
    //Free
    if (M != NULL ){ free(M); }
    if (A != NULL) { free(A) ; }
    if (B != NULL) { free(B) ; }
    if (CudaVecteurA != NULL) { hipFree(CudaVecteurA) ; CudaVecteurA = NULL ; }
    if (CudaVecteurB != NULL) { hipFree(CudaVecteurB) ; CudaVecteurB = NULL ; }
    if (CudaVecteurM != NULL) { hipFree(CudaVecteurM) ; CudaVecteurM = NULL ; }
    if (CudaDiagAy != NULL) { hipFree(CudaDiagAy) ; CudaDiagAy = NULL ; }
    if (CudaDiagBx != NULL) { hipFree(CudaDiagBx) ; CudaDiagBx = NULL ; }
 
   return 0 ; 

}

