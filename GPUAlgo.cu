#include "hip/hip_runtime.h"
#include "MyInc.h" 

// Question 1 en mémoire globale
// Mergesmall_k
__global__ void MergePathGPU_1024(TYPE *A, TYPE *B, TYPE *M, int sizeA, int sizeB)
{
    int i = threadIdx.x ;   
    Point K, P, Q;
    int offset ;

    if (i >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements

    if (i > sizeA) 
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ; 
         Q.x = K.x + offset ; Q.y = K.y - offset ; 

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ; 
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

} // End of MergePathGPU_1024

//Question 1 en mémoire shared 
// Mergesmall_k_shared
__global__ void MergePathGPU_1024_shared(TYPE *GlobalCudaA, TYPE *GlobalCudaB, TYPE *M, int sizeA, int sizeB)
{
    extern __shared__ TYPE dataAB[] ; 
    unsigned int tid = threadIdx.x;
    unsigned int i   = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements

 // Chargement des données dans la mémoire partagée par le thread ; 
     dataAB[tid] = (i < sizeA)?GlobalCudaA[i]:GlobalCudaB[i-sizeA] ; 

 // On attend qur tous les threads aient faits le travail
    __syncthreads(); 

 // On recadre nos pointeurs pourqu'ils pointent vers la mémoire partagée et la globale
    TYPE * A = dataAB ; 
    TYPE * B = dataAB + sizeA ; 

    Point K, P, Q;
    int offset ;

    if (i > sizeA) 
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ; 
         Q.x = K.x + offset ; Q.y = K.y - offset ; 

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ; 
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

} // End of MergePathGPU_1024

//Question 2 
// Pour merge à partir des fenetre obetnue 
// mergeBig k
__global__ void mergeGPU(TYPE * CudaVecteurA, TYPE * CudaVecteurB, TYPE * CudaVecteurC, int * CudaDiagAy, int * CudaDiagBx , int nbthread) 
{ 	
    // int i = threadIdx.x ;     // On renge le Ieme element
    int i = blockIdx.x * blockDim.x + threadIdx.x; // On range le ieme elet 
    int diag = (i / nbthread)  ;   // Dans quel fenêtre est-il ?  
    int indC = nbthread * diag ; 
    
    TYPE *A = CudaVecteurA+CudaDiagAy[diag] ; 
    TYPE *B = CudaVecteurB+CudaDiagBx[diag] ; 
    TYPE *M = CudaVecteurC + indC  ;  
    int sizeA = CudaDiagAy[diag+1]-CudaDiagAy[diag] ; 
    int sizeB = CudaDiagBx[diag+1]-CudaDiagBx[diag] ; 

    Point K, P, Q;
    int offset ;
 
    i = i % nbthread ; // On recadre i dans le nouvel espace
    if (i >= (sizeA + sizeB)) { return ;  }  // On gère les ébordements
    if (i > sizeA) 
    {
       K.x = i - sizeA ; K.y = sizeA ;
       P.x = sizeA ; P.y = i - sizeA ;
    }
    else // x ~ horizontal
    {
       K.x = 0 ; K.y = i ;
       P.x = i ; P.y = 0 ;
    }
    while (1)
    {
         offset = abs(K.y - P.y) / 2 ; 
         Q.x = K.x + offset ; Q.y = K.y - offset ; 

         if ( (Q.y >= 0) && (Q.x <= sizeB) &&
              ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
         {
              if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
              {
                   if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                   {  M[i] = A[Q.y] ; }
                   else
                   {  M[i] = B[Q.x] ; }
                   break ; 
              }
              else
              {  K.x = Q.x + 1 ; K.y = Q.y - 1 ; }
         }
         else
         { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    }

} // End of MergeGPU

// Initialiser les diagonale
__global__ void initDiagGPU(int SizeA , int SizeB, int * CudaDiagBx, int * CudaDiagAy, int NbWindows)  
{ 
   CudaDiagBx[0] = CudaDiagAy[0] = 0 ; 
   CudaDiagBx[NbWindows] = SizeB ;  
   CudaDiagAy[NbWindows] = SizeA ; 
} 

// Question 2
// Pour obtenir les diagolane 
// PathBig k
__global__ void AnalyseDiagonalesGPU(TYPE * CudaVecteurA, TYPE * CudaVecteurB, int sizeA , int sizeB, int * CudaDiagBx, int * CudaDiagAy, int nbthread)  
{
    // int i = blockIdx.x * blockDim.x + threadIdx.x; // On range le ieme elet 
    int nth = threadIdx.x; // On explore le nth diagonale 
    Point K, P, Q ; 
    int   px , py ; 
    TYPE * A = CudaVecteurA ; 
    TYPE * B = CudaVecteurB ; 
    int offset ; 
    int numDiag  = (nth+1) * nbthread -1 ; // Les tableaux vont de 0 à N-1 
	if (numDiag > sizeA) 
        {
    		K.x = numDiag - sizeA ; K.y = sizeA ;
    		P.x = sizeA ; P.y = numDiag - sizeA ;
    	}
    	else // x ~ horizontal
        {
    		K.x = 0 ; K.y = numDiag ;
    		P.x = numDiag ; P.y = 0 ;
    	}
    	while (1)
        {
             offset = abs(K.y - P.y) / 2 ; 
             Q.x = K.x + offset ; Q.y = K.y - offset ; 

             if ( (Q.y >= 0) && (Q.x <= sizeB) &&
                  ( (Q.y == sizeA) || (Q.x == 0) || (A[Q.y] > B[Q.x -1])) )
             {
                  if ((Q.x == sizeB) || (Q.y == 0) || (A[Q.y-1] <= B[Q.x]))
                  {
                       px = Q.x ; py = Q.y ;
                       if ((Q.y < sizeA) && ((Q.x == sizeB) || (A[Q.y] <= B[Q.x])))
                       {  // v = A[Q.y] ; 
                          py ++ ; 
                       }
                       else
                       {  // v = B[Q.x] ; 
                          px ++ ; 
                       }
                       // printf("Analyse Diagonale Point de Sortie ref %d - M %" FMT " Q (A Q.y %d) (B Q.x %d) rv.x %d rv.y %d\n",i,v,Q.y,Q.x,rv->x,rv->y) ; 
                       CudaDiagBx[nth+1] = px ; CudaDiagAy[nth+1] = py ; 
                       break ; // Pour simuler passage au thread suivant
                  }
                  else
                  {  K.x = Q.x + 1 ; K.y = Q.y - 1 ;  }
             }
             else
             { P.x = Q.x -1 ; P.y = Q.y + 1 ; }
    	}
} // End of AnalyseDiagonales
