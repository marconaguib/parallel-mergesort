#include <stdio.h>
#include <stdlib.h>
#include "MyInc.h"
 
#include "main.h" 
 
int main(int argc, char ** argv)
{
   int nbtest = 0 ;
   int SizeA = 800 ;  // Vertical ~ Y
   int SizeB = 200 ;  // Horizontal ~ x
   int nbthread = 512 ;
   int maxTest  =  2 * (int) RemplirEnd ;  
   hipError_t errCuda ;
   
// Info sur la carte
   getInfoCuda() ;

// Hello
    enum EnumMerge AlgoMerge = TriMergePath_1024_shared ; // On choisi le nom de l'algo qu'on veut essayer 
// Rappel :
  //TriMergeSimpleHOST : en séquentiel : algo A 
  //TriMergePathHOST : en séquentiel algo B
  //TriMergePathGPU_1024 : question 1 mémoire globale
  //TriMergePath_1024_shared  : question 1 mémoire shared 
  //TriWindowsGPU_Para : question 2 
  // La question 3 et 5 sont dans des fichiers sépares 


// On fait quelques calculs, même si non utilisés par l'algo cible
   int NbDiagonale  = (SizeA + SizeB) / nbthread ;
   int NbWindows    = NbDiagonale ; 
   // Nombre de trie à faire, c'est le nombre de diagonale +1, sauf si la dernière diagonale est sur 
   // le coin en bas à droite
   NbWindows       += (((SizeA + SizeB) % nbthread) == 0)?0:1 ;  

// Allocation dynamique Sur le host
   allocVecteur(SizeA, SizeB, nbthread)   ; 
 
// Vérifie que la carte et les dimensions sont compatibles avec la carte 
   switch(AlgoMerge)
   {
        case TriMergePathGPU_1024:
             if ((SizeA + SizeB) > 1024)
             { printf("TriMergePathGPU_1024 SizeA + SizeB %d > 1024\n",SizeA + SizeB) ; 
               exit(0) ; 
             }
             break ;
        case TriMergePath_1024_shared:
             if ((SizeA + SizeB) > 1024)
             { printf("TriMergePath_1024_shared SizeA + SizeB %d > 1024\n",SizeA + SizeB) ; 
               exit(0) ; 
             }
             break ;
   }

// Pour chaque génération disponible
    for (int e = 0 ; e < (int)RemplirEnd ; e++)
    {    nbtest ++ ;
         printf("CardA %d - CardB %d - NBThread %d - Algo de remplissage %s - %s\n",
                 SizeA, SizeB, nbthread, MSGRemplir[e],MSGMerge[AlgoMerge]) ;
 
         initVecteur(HostVecteurA, SizeA, HostVecteurB, SizeB, HostVecteurC, (enum EnumTypeRemplissage) e, 1925) ;
 
         if (hipSuccess != (errCuda = hipMemcpy(CudaVecteurA, HostVecteurA, SizeA * sizeof(TYPE), hipMemcpyHostToDevice)))
         { printf("PB copie HostA -> cudaA - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; cleanup() ; exit(2) ; }
 
         if (hipSuccess != (errCuda = hipMemcpy(CudaVecteurB, HostVecteurB, SizeB * sizeof(TYPE), hipMemcpyHostToDevice)))
         { printf("PB copie HostB -> cudaB - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; cleanup() ; exit(2) ; }
 
      // Le resultat est le meme quelque soit l'ordre dans lequel on fait le tri.
         MergeSimpleHOST(HostVecteurA, HostVecteurB, HostVecteurD, SizeA, SizeB) ;
         switch(AlgoMerge)
         {
             case TriMergeSimpleHOST:
                  MergeSimpleHOST(HostVecteurA, HostVecteurB, HostVecteurC, SizeA, SizeB) ; // 1 thread pour 1 grille
             break ;
             case TriMergePathHOST:
                  MergePathHOST(HostVecteurA, HostVecteurB, HostVecteurC, SizeA, SizeB);
             break ;
             case TriMergePathGPU_1024:
                  MergePathGPU_1024<<<1,SizeA+SizeB>>>(CudaVecteurA, CudaVecteurB, CudaVecteurC, SizeA, SizeB) ;
                  if (hipSuccess != (errCuda = hipMemcpy(HostVecteurC, CudaVecteurC,
                                           (SizeA + SizeB) * sizeof(TYPE), hipMemcpyDeviceToHost)))
                  { printf("Error copie cuda C -> host C  - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; 
                    cleanup() ; exit(2) ; 
                  }
	     break ; 
             case TriMergePath_1024_shared: 
                  MergePathGPU_1024_shared<<<1,SizeA+SizeB, (SizeA+SizeB) * sizeof (TYPE)>>>(CudaVecteurA, CudaVecteurB, CudaVecteurC, SizeA, SizeB) ;
                  if (hipSuccess != (errCuda = hipMemcpy(HostVecteurC, CudaVecteurC,
                                           (SizeA + SizeB) * sizeof(TYPE), hipMemcpyDeviceToHost)))
                  { printf("Error copie cuda C -> host C  - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; 
                    cleanup() ; exit(2) ; 
                  }
             break ;
             case TriWindowsGPU_Para:
                  MergeWindowsGPU(CudaVecteurA, CudaVecteurB, CudaVecteurC, SizeA, SizeB , 
                                  CudaDiagAy  , CudaDiagBx , HostDiagAy, HostDiagBx ,  
                                  HostVecteurA , HostVecteurB, HostVecteurC, nbthread, NbDiagonale, NbWindows) ;
                              
                  if (hipSuccess != (errCuda = hipMemcpy(HostVecteurC, CudaVecteurC,
                                           (SizeA + SizeB) * sizeof(TYPE), hipMemcpyDeviceToHost)))
                  { printf("Error copie cuda C -> host C  - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; 
                    cleanup() ; exit(2) ; 
                  }
             break ;
         }

         printf("Verif A versus B %d / %d \n",nbtest, maxTest) ;
         if (compare(HostVecteurD, HostVecteurC, SizeA + SizeB) != 0)
         {   printf("Errorr in %d / %d \n",nbtest, maxTest) ;
             printf("Vect En erreur\n") ;
             //  Affiche((char *)"VectC", HostVecteurC, SizeA+SizeB) ;
             exit(0) ;
         }

         nbtest ++ ; // Pour trier 2 fois A et B, pour vérier que ça marche dans les 2 sens 
         printf("Card First %d - Card Second %d - NBThread %d - Algo de remplissage %s - %s\n",
                 SizeB, SizeA, nbthread, MSGRemplir[e],MSGMerge[AlgoMerge]) ;
         switch(AlgoMerge)
         {
             case TriMergeSimpleHOST:
                  MergeSimpleHOST(HostVecteurB, HostVecteurA, HostVecteurC, SizeB, SizeA) ; // 1 thread pour 1 grille
             break ;
             case TriMergePathHOST:
                  MergePathHOST(HostVecteurB, HostVecteurA, HostVecteurC, SizeB, SizeA);
             break ;
             case TriMergePathGPU_1024:
                  MergePathGPU_1024<<<1,SizeA+SizeB>>>(CudaVecteurB, CudaVecteurA, CudaVecteurC, SizeB, SizeA); 
                  if (hipSuccess != (errCuda = hipMemcpy(HostVecteurC, CudaVecteurC,
                                              (SizeA + SizeB) * sizeof(TYPE), hipMemcpyDeviceToHost)))
                  {  printf("PB copie 2 cuda C -> host C  - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; 
                     cleanup() ; exit(2) ; 
                  }
             break ;
             case TriMergePath_1024_shared: 
                  MergePathGPU_1024_shared<<<1,SizeA+SizeB, (SizeA+SizeB) * sizeof (TYPE)>>>(CudaVecteurB, CudaVecteurA, CudaVecteurC, SizeB, SizeA) ;
                  if (hipSuccess != (errCuda = hipMemcpy(HostVecteurC, CudaVecteurC,
                                           (SizeA + SizeB) * sizeof(TYPE), hipMemcpyDeviceToHost)))
                  { printf("Error copie cuda C -> host C  - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; 
                    cleanup() ; exit(2) ; 
                  }
             break ;
             case TriWindowsGPU_Para:
                  MergeWindowsGPU(CudaVecteurA, CudaVecteurB, CudaVecteurC, SizeA, SizeB , 
                                  CudaDiagAy  , CudaDiagBx , HostDiagAy, HostDiagBx ,  
                                  HostVecteurA , HostVecteurB, HostVecteurC, nbthread, NbDiagonale, NbWindows) ;
                              
                  if (hipSuccess != (errCuda = hipMemcpy(HostVecteurC, CudaVecteurC,
                                           (SizeA + SizeB) * sizeof(TYPE), hipMemcpyDeviceToHost)))
                  {  printf("Error copie cuda C -> host C  - %d - %s\n",errCuda,hipGetErrorName(errCuda)) ; 
                     cleanup() ; exit(2) ; 
                  }
             break ;
        }
 
        printf("Verif B versus A %d / %d \n",nbtest, maxTest) ;
        if (compare(HostVecteurD, HostVecteurC, SizeA + SizeB) != 0)
        {   printf("PHL Erreur in %d - \n",nbtest) ; exit(0) ; }
         
    }
    printf("NB test %d  / %d Pour %s\n",nbtest,  maxTest, MSGMerge[AlgoMerge]) ;

    cleanup() ; printf("Bye Bye\n") ;
 
    return 0 ;
 
}
 
