#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>

// Pour pouvoir experimenter les performances avec les différents types
// FMT  pour print comme on change de type
#define TYPE long
#define FMT  "ld"
#define sizeA 10
#define sizeB 20

// Algorithme A
void MergeSimpleHOST(TYPE *A, TYPE *B, TYPE *M, int cardA, int cardB)
{
    int j = 0;
    int i = 0;

    // On utilise comme pointeurs les arguments de la fonctions
    while (i + j < cardA + cardB )
    {
        if (i >= cardA ) // On a épuisé A, donc on complete avec B
        {  *M = *B; // on utilise les pointeurs pour éviter de faire l opération i+j et se déplacer  = gain de performance
            M = M + 1 ; // Je déplace les pointeurs
            B = B + 1 ;
            j = j + 1 ;
        }
        else if ((j >= cardB) || (*A < *B))
        {   *M = *A ; M = M + 1 ; A = A + 1 ; i = i + 1 ; }
        else
        {   *M = *B ; M = M + 1 ; B = B + 1 ; j = j + 1 ; }
    }
} // End of MergeSimpleHOST

// Algorithme B

// On parle de la notion de point dans l algo B
typedef struct
{
   int x ;
   int y ;
}  Point ;

void MergePathHOST(TYPE *A, TYPE *B, TYPE *M, int cardA, int cardB)
{
    Point K;
    Point P;
    Point Q;
    int offset ;

    for (int i = 0 ; i < cardA + cardB ;  i ++)
    {
    	if (i > cardA)
      {
    		K.x = i - cardA ; K.y = cardA ;
    		P.x = cardA ; P.y = i - cardA ;
    	}
    	else
      {
    		K.x = 0 ; K.y = i ;
    		P.x = i ; P.y = 0 ;
    	}
    	while (1)
      {
        offset = abs (K.y - P.y) / 2 ;
        Q.x = K.x + offset ; Q.y = K.y - offset ; // Q est bien sur une diagonale à 45°

    		if (((Q.y >= 0 ) && (Q.x <= cardB)) &&
    			((Q.y == cardA) || (Q.x == 0) || (A[Q.y]>B[Q.x -1]))){

    			if ((Q.x == cardB) || (Q.y == 0) || (A[Q.y-1]<=B[Q.x]))
          {
    				if((Q.y < cardA) && ((Q.x == cardB) || (A[Q.y] <= B[Q.x])))
            {  M[i]= A[Q.y] ; }
    				else
            {	 M[i] = B[Q.x] ; }
    				break;  // Pour simuler passage au thread suivant
    			}
    			else
          {  K.x = Q.x +1 ; K.y = Q.y - 1 ; }
    		}
    		else
        {	P.x = Q.x -1 ; P.y = Q.y +1 ; }
    	}
    }
}

// Fonctions utiles : permet d afficher le tableau

void Affiche(char * tabMsg, TYPE * ptBuffer, int NB)
{
   TYPE * pt = ptBuffer ;
   for ( int k = 0 ; k < NB  ; k++ , pt ++)
   {   printf(" - %s[%03d] = %6" FMT, tabMsg, k , *pt) ;
       if ((k % 5) == 0) { printf("\n") ; }
   }
   printf("\n") ;
}

// Fonctions utiles : Pour vérifier que le tableau est trié
int check(char * msg, int Nb, TYPE * pto)
{
    TYPE * pt1 = pto ;
    TYPE * pt2 = pto + 1 ;

    for (int i = 0 ; i < Nb-1 ; i ++)
    {
        if (*pt1 > *pt2)
        { printf("Check %s pour %d - Erreur en position %d %"FMT" > %"FMT" \n", msg, Nb, i, *pt1, *pt2) ;

          return i ;
        }
        pt1 ++ ; pt2 ++ ;
    }

    printf("Check %s pour %d est OK \n", msg, Nb) ;
    return 0 ;
}

int main(int argc, char ** argv)
{

   TYPE * vecteurA ;
   TYPE * vecteurB ;
   TYPE * vecteurC ;
   int cas = 1;

// allocation dynamique
   if ((vecteurA = (TYPE *) malloc(sizeA * sizeof(TYPE))) == NULL)
   { printf("PB allocation VecteurA\n") ; exit (1) ; }

   if ((vecteurB = (TYPE *) malloc(sizeB * sizeof(TYPE))) == NULL)
   { printf("PB allocation VecteurB\n") ; exit (1) ; }

   if ((vecteurC = (TYPE *) malloc((sizeA + sizeB) * sizeof(TYPE))) == NULL)
   { printf("PB allocation VecteurC\n") ; exit (1) ; }

// Initialisation des deux vecteurs de base
   if (cas == 1)
   {
       printf("A pair %d B impair %d\n",sizeA,sizeB) ;
       for (int j = 0 ; j < sizeA; j ++) { vecteurA[j] = 2 * j ; }
       for (int j = 0 ; j < sizeB; j ++) { vecteurB[j] = 2 * j + 1 ; }
   }

// vérifier qu'on génére bien les tableaux
   check("Vecteur A ", sizeA, vecteurA) ;
   check("Vecteur B ", sizeB, vecteurB) ;
   Affiche("VectA", vecteurA, sizeA) ;

   //MergeSimpleHOST(vecteurA, vecteurB, vecteurC, sizeA,sizeB) ;
   MergePathHOST(vecteurA, vecteurB, vecteurC, sizeA,sizeB);
   Affiche("VectC", vecteurC, sizeA + sizeB) ;
   check("Vecteur M ",sizeA+sizeB, vecteurC) ;

   return 0 ;
}
